#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <cmath>
#include <algorithm>
#include <string>
#include <ctime>
#include <chrono>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>	
#include <thrust/transform.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

#include <TObject.h>
#include <TROOT.h>
#include <TFile.h>
#include <TTree.h>
#include <TRandom.h>
#include <TMatrixD.h>
#include <TLorentzVector.h>
#include <TClonesArray.h>
#include <TStopwatch.h>
#include <TTimeStamp.h>
#include <TString.h>
#include "LoadInput.h"

#define nChamberPlanes 30
#define nHodoPlanes 16
#define nPropPlanes 8

#define triggerBit(n) (1 << (n))
#define hitFlagBit(n) (1 << (n))

using namespace std;

const int EstnEvtMax = 10240;
const int THREADS_PER_BLOCK = 512;
int BLOCKS_NUM = EstnEvtMax/THREADS_PER_BLOCK;
const int EstnAHMax = 5000;
const int EstnTHMax = 200;
const int ClusterSizeMax = 100;

class gHit {
	public:
	int index;
	short detectorID;
	short elementID;
	float tdcTime;
	float driftDistance;
	float pos;
	short flag;
	
};

class gEvent {
	public:
	int RunID;
	int EventID;
	int SpillID;
	int TriggerBits;
	short TargetPos;
	int TurnID;
	int RFID;
	int Intensity[33];
	short TriggerEmu;
	short NRoads[4];
	int NHits[nChamberPlanes+nHodoPlanes+nPropPlanes+1];
	int nAH;
	int nTH;
	gHit AllHits[EstnAHMax];
	gHit TriggerHits[EstnTHMax];
	 
          
};

struct lessthan {
	__host__ __device__ bool operator()(const gHit& lhs, const gHit& rhs)
	{
		if(lhs.detectorID < rhs.detectorID)
		{
			return true;
		}
		else if(lhs.detectorID > rhs.detectorID)
		{
			return false;
		}

		if(lhs.elementID < rhs.elementID)
		{
			return true;
		}
		else if(lhs.elementID > rhs.elementID)
		{
			return false;
		}

		if(lhs.tdcTime > rhs.tdcTime)
		{
			return true;
		}
		else
		{
			return false;
		}
	}
};

__global__ void gkernel(gEvent* ic, int* og) {

	   
	   
	   //printf("Running the kernel function...\n");
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double w_max[EstnEvtMax];
	double w_min[EstnEvtMax];
	double dt_mean[EstnEvtMax];
	int cluster_iAH_arr_cur[EstnEvtMax];
	int cluster_iAH_arr_size[EstnEvtMax];
	static int cluster_iAH_arr[EstnEvtMax][ClusterSizeMax];
	int uniqueID[EstnEvtMax];
	int uniqueID_curr[EstnEvtMax];
	double tdcTime_curr[EstnEvtMax];
	int iAH[EstnEvtMax];
	int nAH_reduced[EstnEvtMax];
	//int reduced_Event[EstnEvtMax];      

          
	  
 
	
	cluster_iAH_arr_size[index] = 0;
	nAH_reduced[index] = 0;
	for(iAH[index] = 0; iAH[index]<ic[index].nAH; ++iAH[index]) {
		if((ic[index].AllHits[iAH[index]].flag & hitFlagBit(1)) == 0) {
//			printf("Skip out-of-time...\n");
			ic[index].AllHits[iAH[index]].detectorID = 0;
			continue;
		}
		if(ic[index].AllHits[iAH[index]].detectorID < 31 || ic[index].AllHits[iAH[index]].detectorID > 46) {
			uniqueID[index] = ic[index].AllHits[iAH[index]].detectorID*1000 + ic[index].AllHits[iAH[index]].elementID;
			if(uniqueID[index] != uniqueID_curr[index]) {
				uniqueID_curr[index] = uniqueID[index];
				tdcTime_curr[index] = ic[index].AllHits[iAH[index]].tdcTime;
			}
			else {
				if(ic[index].AllHits[iAH[index]].detectorID > 36 || ((ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] >= 0.0) && (ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] < 80.0)) || ((ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] <= 0.0) && (ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] > -80.0))) {
//					printf("Skip after-pulse...\n");
					ic[index].AllHits[iAH[index]].detectorID = 0;
					continue;
				}
				else {
					tdcTime_curr[index] = ic[index].AllHits[iAH[index]].tdcTime;
					
				}
			}
		}
		if(ic[index].AllHits[iAH[index]].detectorID <= nChamberPlanes) {
//			printf("%d\n", cluster_iAH_arr_size[index]);
//			printf("Decluster...\n");
			if(cluster_iAH_arr_size[index] == ClusterSizeMax) {
//				printf("Oversized cluster...\n");
			}
			if(cluster_iAH_arr_size[index] == 0) {
				cluster_iAH_arr[index][0] = iAH[index];
				++cluster_iAH_arr_size[index];
			}
			else {
				if((ic[index].AllHits[iAH[index]].detectorID != ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID) || (ic[index].AllHits[iAH[index]].elementID - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].elementID > 1)) {
					if(cluster_iAH_arr_size[index] == 2) {
						w_max[index] = 0.9*0.5*(ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].pos - ic[index].AllHits[cluster_iAH_arr[index][0]].pos);
						w_min[index] = 4.0/9.0*w_max[index];
						if((ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > w_max[index] && ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance > w_min[index]) || (ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > w_min[index] && ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance > w_max[index])) {
							if(ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance) {
//								printf("Skip cluster...\n");
								ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID = 0;
							}
							else {
//								printf("Skip cluster...\n");
         
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID = 0;
							}
						}
						else if((((ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) >= 0.0 && (ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) < 8.0) || ((ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) <= 0.0 && (ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) > -8.0)) && (ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID >= 19 && ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID <= 24)) {
//							printf("Skip cluster...\n");
							ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID = 0;
							ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID = 0;
						}
					}
					if(cluster_iAH_arr_size[index] >= 3) {
						dt_mean[index] = 0.0;
						for(cluster_iAH_arr_cur[index] = 1; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
							dt_mean[index] += ((ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime) > 0.0 ? (ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime) : (ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime));
						}
						dt_mean[index] = dt_mean[index]/(cluster_iAH_arr_size[index] - 1);
						if(dt_mean[index] < 10.0) {
//							printf("Skip cluster...\n");
							for(cluster_iAH_arr_cur[index] = 0; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].detectorID = 0;
							}
						}
						else {
//							printf("Skip cluster...\n");
							for(cluster_iAH_arr_cur[index] = 1; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].detectorID = 0;
							}
						}
					}
					cluster_iAH_arr_size[index] = 0;
				}
				cluster_iAH_arr[index][cluster_iAH_arr_size[index]] = iAH[index];
				++cluster_iAH_arr_size[index];
			}
		}
	}

	for(iAH[index] = 0; iAH[index]<ic[index].nAH; ++iAH[index]) {
		if(ic[index].AllHits[iAH[index]].detectorID != 0) {
			ic[index].AllHits[nAH_reduced[index]] = ic[index].AllHits[iAH[index]];
			++nAH_reduced[index];
			
		}
	}

	ic[index].nAH = nAH_reduced[index];
	//reduced_Event[index] = ic[index].EventID;
	//if( ((reduced_Event[index])<6668 )) {
          //printf("EventID: %d\n", (ic[index].AllHits[nAH_reduced[index]]) );
          //printf("EventID: %d\n", (ic[index].nAH) );
        
//}		     
	
	//if( ((ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])<1) || ((ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])>350) || ((ic[index].NHits[7]+ic[index].NHits[8]+ic[index].NHits[9]+ic[index].NHits[10]+ic[index].NHits[11]+ic[index].NHits[12])>350) || ((ic[index].NHits[13]+ic[index].NHits[14]+ic[index].NHits[15]+ic[index].NHits[16]+ic[index].NHits[17]+ic[index].NHits[18])>170) || ((ic[index].NHits[19]+ic[index].NHits[20]+ic[index].NHits[21]+ic[index].NHits[22]+ic[index].NHits[23]+ic[index].NHits[24])>140) || ((ic[index].NHits[25]+ic[index].NHits[26]+ic[index].NHits[27]+ic[index].NHits[28]+ic[index].NHits[29]+ic[index].NHits[30])>140)){

	if( (ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])>1)
	    {
	    //printf("EventID: %d\n", (ic[index].EventID));ic[index].AllHits[nAH_reduced[index]]
	    //printf("D0 : %d\n", (ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6]));
		//printf("Event rejected...\n");
		//printf((ic[index].EventID));
	}
	else {

	
	printf("Reduced hits: %d\n", (nAH_reduced[index]) );
	
	//if( ((reduced_Event[index])<6668 )) {	  
	  
	  //printf("D0 : %d\n", (ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6]));
	  //printf("D1 : %d\n", (ic[index].NHits[7]+ic[index].NHits[8]+ic[index].NHits[9]+ic[index].NHits[10]+ic[index].NHits[11]+ic[index].NHits[12]));
	  //printf("D2 : %d\n", (ic[index].NHits[13]+ic[index].NHits[14]+ic[index].NHits[15]+ic[index].NHits[16]+ic[index].NHits[17]+ic[index].NHits[18]));
	  //printf("D3p : %d\n", (ic[index].NHits[19]+ic[index].NHits[20]+ic[index].NHits[21]+ic[index].NHits[22]+ic[index].NHits[23]+ic[index].NHits[24]));
	  //printf("D3m : %d\n", (ic[index].NHits[25]+ic[index].NHits[26]+ic[index].NHits[27]+ic[index].NHits[28]+ic[index].NHits[29]+ic[index].NHits[30]));
	  //printf("H1 : %d\n", (ic[index].NHits[31]+ic[index].NHits[32]));
	  //printf("H2 : %d\n", (ic[index].NHits[33]+ic[index].NHits[34]));
	  //printf("H3 : %d\n", (ic[index].NHits[35]+ic[index].NHits[36]));
	  //printf("H1 : %d\n", (ic[index].NHits[37]+ic[index].NHits[38]));
	  //printf("prop : %d\n", (ic[index].NHits[47]+ic[index].NHits[48]+ic[index].NHits[49]+ic[index].NHits[50]));
	  //printf("prop : %d\n", (ic[index].NHits[51]+ic[index].NHits[52]+ic[index].NHits[53]+ic[index].NHits[54]));
	  //printf("prop : %d\n", (ic[index].NHits[55]));
	  //printf("prop : %d\n", (ic[index].NHits[56]));
	  //printf("prop : %d\n", (ic[index].NHits[57]));
	  //printf("prop : %d\n", (ic[index].NHits[58]));
	  //printf("prop : %d\n", (ic[index].NHits[59]));
	  
	    //}
	    
//		Process the accepted events (tracking) here.
		}

}



int main(int argc, char* argv[]) {

	auto start = std::chrono::system_clock::now();
	clock_t cp1 = clock();

	TString inputFile;
	TString outputFile;
	inputFile = argv[1];
	outputFile = argv[2];

	cout<<"Running "<<argv[0]<<endl;
	cout<<"Loading "<<argv[1]<<endl;
	cout<<"Writing "<<argv[2]<<endl;

	SRawEvent* rawEvent = new SRawEvent();
	TFile* dataFile = new TFile(inputFile.Data(), "READ");
	TTree* dataTree = (TTree *)dataFile->Get("save");
	dataTree->SetBranchAddress("rawEvent", &rawEvent);
	int nEvtMax = dataTree->GetEntries();
	static gEvent host_gEvent[EstnEvtMax];



	for(int i = 0; i < nEvtMax; ++i) {
		dataTree->GetEntry(i);
		//cout<<"Converting "<<i<<"/"<<nEvtMax<<endl;

		host_gEvent[i].RunID = rawEvent->fRunID;
		host_gEvent[i].EventID = rawEvent->fEventID;
		host_gEvent[i].SpillID = rawEvent->fSpillID;
		host_gEvent[i].TriggerBits = rawEvent->fTriggerBits;
		host_gEvent[i].TargetPos = rawEvent->fTargetPos;
		host_gEvent[i].TurnID = rawEvent->fTurnID;
		host_gEvent[i].RFID = rawEvent->fRFID;
		for(int j=0; j<33; j++) {
			host_gEvent[i].Intensity[j] = rawEvent->fIntensity[j];
		}
		host_gEvent[i].TriggerEmu = rawEvent->fTriggerEmu;
		for(int k=0; k<4; k++) {
			host_gEvent[i].NRoads[k] = rawEvent->fNRoads[k];
		}
		for(int l=0; l<(nChamberPlanes+nHodoPlanes+nPropPlanes+1); l++) {
			host_gEvent[i].NHits[l] = rawEvent->fNHits[l];
		}
		host_gEvent[i].nAH = rawEvent->fAllHits.size();
		host_gEvent[i].nTH = rawEvent->fTriggerHits.size();
		for(int m=0; m<rawEvent->fAllHits.size(); m++) {
			host_gEvent[i].AllHits[m].index=(rawEvent->fAllHits[m]).index;
			host_gEvent[i].AllHits[m].detectorID=(rawEvent->fAllHits[m]).detectorID;
			host_gEvent[i].AllHits[m].elementID=(rawEvent->fAllHits[m]).elementID;
			host_gEvent[i].AllHits[m].tdcTime=(rawEvent->fAllHits[m]).tdcTime;
			host_gEvent[i].AllHits[m].driftDistance=(rawEvent->fAllHits[m]).driftDistance;
			host_gEvent[i].AllHits[m].pos=(rawEvent->fAllHits[m]).pos;
			host_gEvent[i].AllHits[m].flag=(rawEvent->fAllHits[m]).flag;

              


}
		for(int n=0; n<rawEvent->fTriggerHits.size(); n++) {
			host_gEvent[i].TriggerHits[n].index=(rawEvent->fTriggerHits[n]).index;
			host_gEvent[i].TriggerHits[n].detectorID=(rawEvent->fTriggerHits[n]).detectorID;
			host_gEvent[i].TriggerHits[n].elementID=(rawEvent->fTriggerHits[n]).elementID;
			host_gEvent[i].TriggerHits[n].tdcTime=(rawEvent->fTriggerHits[n]).tdcTime;
			host_gEvent[i].TriggerHits[n].driftDistance=(rawEvent->fTriggerHits[n]).driftDistance;
			host_gEvent[i].TriggerHits[n].pos=(rawEvent->fTriggerHits[n]).pos;
			host_gEvent[i].TriggerHits[n].flag=(rawEvent->fTriggerHits[n]).flag;
		}
	}

 
		
//If the decoded has NOT been sorted...
//	for(int i = 0; i < nEvtMax; ++i) {
//		thrust::stable_sort(host_gEvent[i].AllHits, host_gEvent[i].AllHits+host_gEvent[i].nAH, lessthan());
//	}


	
	gEvent *device_gEvent;
	int sizeofRaw = EstnEvtMax*sizeof(gEvent);

	
	int *host_output[EstnEvtMax];
	int *device_output;
	int sizeofoutput = EstnEvtMax*sizeof(int);
	

	clock_t cp2 = clock();
	auto start_kernel = std::chrono::system_clock::now();

	
	hipMalloc((void**)&device_gEvent, sizeofRaw);
	hipMalloc((void**)&device_output, sizeofoutput);

	//for(i = 0, i < EstnEvtMax; ++i) host_gEvent[i] = 100.f+i;


	hipMemcpy(device_gEvent, host_gEvent, sizeofRaw, hipMemcpyHostToDevice);
	hipMemcpy(device_output, host_output, sizeofoutput, hipMemcpyHostToDevice);

	gkernel<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent, device_output);

	//ofstream output;
        //output.open("reduced_Event.txt");

	//output << "device event:" << (device_output);
	 //for(int i = 0; i < EstnEvtMax; ++i){
        //output <<"Reduced events total: " << "9605" << "\n" << "\n" << "CPU time: 1.38 " << "\n" << "GPU time: 0.88 ";
        //output <<"Reduced_eventID: " << (host_output[i]) << "\n" << "count: " << i << "\n";
        //output<<"count: "<<i;
        //}
	 //output.close();


	hipMemcpy(host_gEvent, device_gEvent, sizeofRaw, hipMemcpyDeviceToHost);
	hipMemcpy(host_output, device_output, sizeofoutput, hipMemcpyDeviceToHost);

	

	//for(int i = 0; i < EstnEvtMax; ++i){
                //cout<<"Input: "<< (host_gEvent[i]) <<endl;
		//cout<<"Output: "<< (host_output[i]) <<endl;
		//cout<<"Input: "<< (host_gEvent[i]) <<endl;
		//}


	hipFree(device_gEvent);
	hipFree(device_output);

	auto end_kernel = std::chrono::system_clock::now();
	clock_t cp3 = clock();

	delete rawEvent;

	
	//ofstream output;
	//output.open("reduced_Event.txt");
	
	//for(int i = 0; i < 10000; ++i){
	//output <<"Reduced events total: " << "9605" << "\n" << "\n" << "CPU time: 1.38 " << "\n" << "GPU time: 0.88 ";
	//output <<"Reduced_eventID: " << (host_output[i]) << "\n" << "count: " << i << "\n";
	//outp0ut<<"count: "<<i;
	//}
	
	//output.close();
	
	
	//for(int i = 0; i < 55; ++i) {
		//cout<<"output: "<<(host_gEvent[6667].NHits[i])<<endl;
		//cout<<"output: "<<(host_gEvent[i])<<endl;
		//cout<<"output: "<<(device_gEvent)<<endl;
		//cout<<"output: "<<(sizeof(int))<<endl;
		//cout<<"size: "<<i<<endl;
	//}

	clock_t cp4 = clock();
	auto end = std::chrono::system_clock::now();

	double cpu_secs = double(cp4-cp3+cp2-cp1) / CLOCKS_PER_SEC;
	auto gpu_ns = end_kernel - start_kernel;
	auto overall = end - start;
	cout<<"CPU time: "<<cpu_secs<<endl;
	cout<<"GPU time: "<<(gpu_ns.count()/1000000000.0)<<endl;
	cout<<"Total time: "<<(overall.count()/1000000000.0)<<endl;

	return 0;
}

//e906-gat2:/seaquest/users/hjiang/online_reconstruction
