#include "hip/hip_runtime.h"
#include "gpufit.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <fstream>
#include <string>
#include <ctime>
#include <chrono>

// CUDA runtime
// #include <hip/hip_runtime.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

// #include <hipblaslt.h>

#include <TObject.h>
#include <TROOT.h>
#include <TFile.h>
#include <TTree.h>
#include <TRandom.h>
#include <TMatrixD.h>
#include <TLorentzVector.h>
#include <TClonesArray.h>
#include <TStopwatch.h>
#include <TTimeStamp.h>
#include <TString.h>
#include <unordered_map>
//#include "LoadInput.h"
#include "OROutput.h"

#define nChamberPlanes 30
#define nHodoPlanes 16
#define nPropPlanes 8
#define nDetectors (nChamberPlanes+nHodoPlanes+nPropPlanes)
#define Epsilon 0.00001f

#define triggerBit(n) (1 << (n))
#define hitFlagBit(n) (1 << (n))

using namespace std;

const int EstnEvtMax = 10240;
const int THREADS_PER_BLOCK = 512;
int BLOCKS_NUM = EstnEvtMax/THREADS_PER_BLOCK;
const int EstnAHMax = 5000;
const int EstnTHMax = 200;
const int ClusterSizeMax = 100;
const int TrackletSizeMax = 200;// guess

const int TX_MAX = 0.15;
const int TY_MAX = 0.1;
const int X0_MAX = 150;
const int Y0_MAX = 50;
const int INVP_MAX = 0.2;
const int INVP_MIN = 0.01;


typedef std::unordered_map<int, double>::value_type   posType;

// function to check GPU status
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// looping on all GPUs to check their status
void printDeviceStatus() {
	int nDevices;

	
	gpuErrchk( hipGetDeviceCount(&nDevices) );
	for (int i = 0; i < nDevices; i++) {
	  hipDeviceProp_t prop;
	  hipGetDeviceProperties(&prop, i);
	  printf("Device Number: %d\n", i);
	  printf("  Device name: %s\n", prop.name);
	  printf("  Memory Clock Rate (KHz): %d\n",
			 prop.memoryClockRate);
	  printf("  Memory Bus Width (bits): %d\n",
			 prop.memoryBusWidth);
	  printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}
}





//clone of LoadEvent::Hit:
class gHit {
	public:
	int index; // global hit index in the hit array
	short detectorID; // ID of the detector: one ID for each DC wire plane (30 total), hodoscope plane (16 total), proportional tube plane (8 total).
	short elementID; // ID of the element in the detector: wire/slat/tube number
	float tdcTime; // raw TDC time from the DAQ 
	float driftDistance; // calculated drift distance from RT profile (supplied in database) IF tdcTime between tmin and tmax defined for detector; 
	float pos; // position in the projection of the detector (e.g. X in a X plane, etc)
	short flag; // 1: in time; 2: hodo mask; 3: trigger mask
};

class gTracklet {
      public:
      gTracklet(){
	nXHits = nUHits = nVHits = 0;
      }
      
      int stationID;
      int nXHits;
      int nUHits;
      int nVHits;

      float chisq;
      float chisq_vtx;

      gHit hits[nDetectors];// array of all hits

      float tx;
      float ty;
      float x0;
      float y0;
      float invP;
      
      float err_tx;
      float err_ty;
      float err_x0;
      float err_y0;
      float err_invP;
      
      float residual[nChamberPlanes];
};

class gEvent {
	public:
	int RunID; // Run Number
	int EventID; // Event number
	int SpillID; // Spill number
	int TriggerBits; // hash of the trigger bits: 0-4: MATRIX1-5; 5-9: NIM1-5;
	short TargetPos; // target position: proxy for target ID?
	int TurnID; // => related to beam intensity
	int RFID; // => related to beam intensity
	int Intensity[33]; //  16 before, one onset, and 16 after
	short TriggerEmu; // 1 if MC event
	short NRoads[4]; // 0, positive top; 1, positive bottom; 2, negative top; 3, negative bottom
	int NHits[nDetectors+1]; // number of hits in each detector plane
	int nAH; // size of AllHits
	int nTH; // size of TriggerHits
	gHit AllHits[EstnAHMax]; // array of all hits
	gHit TriggerHits[EstnTHMax]; // array of trigger hits
};

//Output class
class gSW {
public:
	int EventID;
	int nAH;
	int nTracklets;
	gTracklet AllTracklets[TrackletSizeMax];//add tracklet here???
};

//geometry carrier
class gPlane {
      public:
      float z;
      int nelem;
      float spacing;
      float xoffset;
      float scalex;
      float x0;
      float costheta;
      float scaley;
      float y0;
      float sintheta;
      float deltaW_[9];
      float z_mean;
      float u_win;
      float v_win_fac1;
      float v_win_fac2;
      float v_win_fac3;
};

// Hit comparison
struct lessthan {
	__host__ __device__ bool operator()(const gHit& lhs, const gHit& rhs)
	{
	//returns true if :
	// hit1.detID<hit2.detID;  
		if(lhs.detectorID < rhs.detectorID)
		{
			return true;
		}
		else if(lhs.detectorID > rhs.detectorID)
		{
			return false;
		}
	//hit1.detID=hit2.detID & hit1.elID<hit2.elID;
		if(lhs.elementID < rhs.elementID)
		{
			return true;
		}
		else if(lhs.elementID > rhs.elementID)
		{
			return false;
		}
	//hit1.detID=hit2.detID & hit1.elID=hit2.elID & hit1.time>hit2.time;
		if(lhs.tdcTime > rhs.tdcTime)
		{
			return true;
		}
		else
		{
			return false;
		}
	}
};


// position of first hit 
struct get_first_event_hit_pos
{
  __host__ __device__
  float operator()(gEvent& evt)
  {
    return evt.AllHits[0].pos;
  }
};

// Linear regression: fit of tracks
void linear_regression_example(int n_points_per_fit, REAL *device_input, thrust::device_vector<REAL> &d_parameters)
{
	// number of fits, fit points and parameters
	size_t const n_fits = 1;
	size_t const n_model_parameters = 2;


	// custom z positions, stored in user info
	// NOTE: this is the way to initialize a device_vector with many constant values
	vector< REAL > user_info_values {-130.43f, -131.437f, -116.782f, -115.712f, -128.114f, -129.111f };
	thrust::device_vector<REAL> d_user_info(user_info_values.size());
	thrust::copy(user_info_values.begin(), user_info_values.end(), d_user_info.begin());

	// size of user info in bytes
	size_t const user_info_size = d_user_info.size() * sizeof(REAL); 

	// // initial parameters (randomized)
	// std::vector< REAL > initial_parameters(n_fits * n_model_parameters);
	// for (size_t i = 0; i != n_fits; i++)
	// {
	// 	// random offset
	// 	initial_parameters[i * n_model_parameters + 0] = true_parameters[0] * (0.8f + 0.4f * uniform_dist(rng));
	// 	// random slope
	// 	initial_parameters[i * n_model_parameters + 1] = true_parameters[0] * (0.8f + 0.4f * uniform_dist(rng));
	// }

	// // generate data
	// std::vector< REAL > data(n_points_per_fit * n_fits);
	// for (size_t i = 0; i != data.size(); i++)
	// {

	// 	size_t j = i / n_points_per_fit; // the fit
	// 	size_t k = i % n_points_per_fit; // the position within a fit

	// 	REAL x = user_info[k];
	// 	REAL y = true_parameters[0] + x * true_parameters[1];
	// 	data[i] = y;



	// }

	// tolerance
	REAL const tolerance = 0.001f;

	// maximum number of iterations
	int const max_number_iterations = 20;

	// estimator ID
	int const estimator_id = LSE;

	// model ID
	int const model_id = LINEAR_1D;

	// parameters to fit (all of them)
	std::vector< int > parameters_to_fit(n_model_parameters, 1);

	
	thrust::device_vector< int > d_states(n_fits);
	thrust::device_vector< REAL > d_chi_square(n_fits);
	thrust::device_vector< int > d_number_iterations(n_fits);

	//call to gpufit (C interface)
	// can be found in https://github.com/gpufit/Gpufit/blob/master/Gpufit/gpufit.cpp
	// parameters:
	// size_t n_fits,
    	// size_t n_points,
	// float * gpu_data,
	// float * gpu_weights,
    	// int model_id,
    	// float tolerance,
    	// int max_n_iterations,
    	// int * parameters_to_fit,
    	// int estimator_id,
	// size_t user_info_size,
    	// char * gpu_user_info,
    	// float * gpu_fit_parameters,
    	// int * gpu_output_states,
    	// float * gpu_output_chi_squares,
    	// int * gpu_output_n_iterations
	// size_t n_fits,

	int const status = gpufit_cuda_interface
       (
        	n_fits,
        	n_points_per_fit,
            device_input,
            0,
            model_id,
            // initial_parameters.data(),
            tolerance,
            max_number_iterations,
            parameters_to_fit.data(),
			// true_parameters.data(),
            estimator_id,
            user_info_size,
            reinterpret_cast< char * >( thrust::raw_pointer_cast(d_user_info.data()) ),
            thrust::raw_pointer_cast(d_parameters.data()),
            thrust::raw_pointer_cast(d_states.data()),
            thrust::raw_pointer_cast(d_chi_square.data()),
            thrust::raw_pointer_cast(d_number_iterations.data())
        );

		

	// check status
	if (status != ReturnState::OK)
	{
		throw std::runtime_error(gpufit_get_last_error());
	}

	// get fit states
	std::vector< int > output_states_histogram(5, 0);
	for (auto it = d_states.begin(); it != d_states.end(); ++it)
	{
		output_states_histogram[*it]++;
	}

	std::cout << "ratio converged              " << (REAL) output_states_histogram[0] / n_fits << "\n";
	std::cout << "ratio max iteration exceeded " << (REAL) output_states_histogram[1] / n_fits << "\n";
	std::cout << "ratio singular hessian       " << (REAL) output_states_histogram[2] / n_fits << "\n";
	std::cout << "ratio neg curvature MLE      " << (REAL) output_states_histogram[3] / n_fits << "\n";
	std::cout << "ratio gpu not read           " << (REAL) output_states_histogram[4] / n_fits << "\n";

	// compute mean fitted parameters for converged fits
	std::vector< REAL > output_parameters_mean(n_model_parameters, 0);
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			// add offset
			output_parameters_mean[0] += d_parameters[i * n_model_parameters + 0];
			// add slope
			output_parameters_mean[1] += d_parameters[i * n_model_parameters + 1];
		}
	}
	output_parameters_mean[0] /= output_states_histogram[0];
	output_parameters_mean[1] /= output_states_histogram[0];

	// compute std of fitted parameters for converged fits
	std::vector< REAL > output_parameters_std(n_model_parameters, 0);
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			// add squared deviation for offset
			output_parameters_std[0] += (d_parameters[i * n_model_parameters + 0] - output_parameters_mean[0]) * (d_parameters[i * n_model_parameters + 0] - output_parameters_mean[0]);
			// add squared deviation for slope
			output_parameters_std[1] += (d_parameters[i * n_model_parameters + 1] - output_parameters_mean[1]) * (d_parameters[i * n_model_parameters + 1] - output_parameters_mean[1]);
		}
	}
	// divide and take square root
	output_parameters_std[0] = sqrt(output_parameters_std[0] / output_states_histogram[0]);
	output_parameters_std[1] = sqrt(output_parameters_std[1] / output_states_histogram[0]);

	// print mean and std
	std::cout << "offset  true " << d_parameters[0] << " mean " << output_parameters_mean[0] << " std " << output_parameters_std[0] << "\n";
	std::cout << "slope   true " << d_parameters[1] << " mean " << output_parameters_mean[1] << " std " << output_parameters_std[1] << "\n";

	// compute mean chi-square for those converged
	REAL  output_chi_square_mean = 0;
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			output_chi_square_mean += d_chi_square[i];
		}
	}
	output_chi_square_mean /= static_cast<REAL>(output_states_histogram[0]);
	std::cout << "mean chi square " << output_chi_square_mean << "\n";

	// compute mean number of iterations for those converged
	REAL  output_number_iterations_mean = 0;
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			output_number_iterations_mean += static_cast<REAL>(d_number_iterations[i]);
		}
	}

	// normalize
	output_number_iterations_mean /= static_cast<REAL>(output_states_histogram[0]);
	std::cout << "mean number of iterations " << output_number_iterations_mean << "\n";
}


// kernel functions: 
// CUDA C++ extends C++ by allowing the programmer to define C++ functions, called kernels, that, when called, 
// are executed N times in parallel by N different CUDA threads, as opposed to only once like regular C++ functions. 

// event reducer: 
__global__ void gkernel_eR(gEvent* ic) {
	//printf("Running the kernel function...\n");
	// retrieve global thread index
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double w_max[EstnEvtMax]; // max drift distance of the hit furthest from the cluster avg position // current average position of cluster * 0.9
	
	double w_min[EstnEvtMax]; // max drift distance of the hit closest to the cluster avg position // current average position of cluster * 0.4
	double dt_mean[EstnEvtMax]; // tbd
	int cluster_iAH_arr_cur[EstnEvtMax]; // current cluster array
	int cluster_iAH_arr_size[EstnEvtMax]; // cluster size i.e number of hits in cluster
	static int cluster_iAH_arr[EstnEvtMax][ClusterSizeMax]; // global cluster array 
	int uniqueID[EstnEvtMax]; // hit unique element ID
	int uniqueID_curr[EstnEvtMax]; // current hit unique element ID
	double tdcTime_curr[EstnEvtMax]; // current hit TDC time
	int iAH[EstnEvtMax]; // hit index 
	int nAH_reduced[EstnEvtMax]; // number of hits after hit quality filtering
	// int nHitsPerDetector[nDetectors+1];
	

	// initialization of array size
	cluster_iAH_arr_size[index] = 0;
	nAH_reduced[index] = 0;
		
	// event reducing/hit filtering
	for(iAH[index] = 0; iAH[index]<ic[index].nAH; ++iAH[index]) {
		// if hit not good, set its detID to 0 and continue;
		if((ic[index].AllHits[iAH[index]].flag & hitFlagBit(1)) == 0) {
//			printf("Skip out-of-time...\n");
			ic[index].AllHits[iAH[index]].detectorID = 0;
			continue;
		}
		// hits in DCs or Prop tubes
		if(ic[index].AllHits[iAH[index]].detectorID < 31 || ic[index].AllHits[iAH[index]].detectorID > 46) {
			// evaluate "unique ID"
			uniqueID[index] = ic[index].AllHits[iAH[index]].detectorID*1000 + ic[index].AllHits[iAH[index]].elementID;
			// compare with current unique element ID; if different, update the unique element ID and time info 
			if(uniqueID[index] != uniqueID_curr[index]) {
				uniqueID_curr[index] = uniqueID[index];
				tdcTime_curr[index] = ic[index].AllHits[iAH[index]].tdcTime;
			}
			// if next hit and current hit belong to the same element: if detID>36 => prop tubes (reminder that hodoscpes are out of the picture in this scope), 
			// we're suppose to have one signal (a second signal would be after-pulsing)
			// if time difference between new hit and current hit is less than 80ns for DCs, it's also considered after-pulsing
			else {
				if(ic[index].AllHits[iAH[index]].detectorID > 36 || ((ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] >= 0.0) && (ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] < 80.0)) || ((ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] <= 0.0) && (ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] > -80.0))) {
//					printf("Skip after-pulse...\n");
					ic[index].AllHits[iAH[index]].detectorID = 0;
					continue;
				}
				else {
					tdcTime_curr[index] = ic[index].AllHits[iAH[index]].tdcTime;
				}
			}
		}
		// declustering of hits in DCs (from CPU code, I understand this one better)
		// if there are hits in the same plane and hitting two neighboring wires, they both give redundant information: 
		if(ic[index].AllHits[iAH[index]].detectorID <= nChamberPlanes) {
//			printf("%d\n", cluster_iAH_arr_size[index]);
//			printf("Decluster...\n");
			if(cluster_iAH_arr_size[index] == ClusterSizeMax) {
//				printf("Oversized cluster...\n");
			}
			// if array size is zero, start storing the hit in the array
			if(cluster_iAH_arr_size[index] == 0) {
				cluster_iAH_arr[index][0] = iAH[index];
				++cluster_iAH_arr_size[index];
			}
			// otherwise
			else {
				// current hit and previous hit are *not* in same detector plane OR next hit and current hit are *not* in neighbors cells
				// we "declusterize" i.e. we remove the hit/hits which information is redundant with other hits and/or useless
				if((ic[index].AllHits[iAH[index]].detectorID != ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID) || (ic[index].AllHits[iAH[index]].elementID - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].elementID > 1)) {
					// if 2 hits in cluster, evaluate w_max and w_min; drift distance has to be < w_min for one of the hits, while it has to be < w_max for the other hit 
					if(cluster_iAH_arr_size[index] == 2) {
						w_max[index] = 0.9*0.5*(ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].pos - ic[index].AllHits[cluster_iAH_arr[index][0]].pos);
						w_min[index] = 4.0/9.0*w_max[index];
						if((ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > w_max[index] && ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance > w_min[index]) || (ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > w_min[index] && ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance > w_max[index])) {
							//eliminating the existing hit with the lagest drift distance
							if(ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance) {
//								printf("Skip cluster...\n");
								ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID = 0;
							}
							else {
//								printf("Skip cluster...\n");
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID = 0;
							}
						}
						// if the time difference is less than 8 ns for detectors 19 to 24 (which btw are DC3p) we remove both
						else if((((ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) >= 0.0 && (ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) < 8.0) || ((ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) <= 0.0 && (ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) > -8.0)) && (ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID >= 19 && ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID <= 24)) {
//							printf("Skip cluster...\n");
							ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID = 0;
							ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID = 0;
						}
					}
					// if 3 hits or more in cluster: we essentially discard them all;
					if(cluster_iAH_arr_size[index] >= 3) {
						// evaluate the mean time difference;
						dt_mean[index] = 0.0;
						for(cluster_iAH_arr_cur[index] = 1; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
							dt_mean[index] += ((ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime) > 0.0 ? (ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime) : (ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime));
						}
						dt_mean[index] = dt_mean[index]/(cluster_iAH_arr_size[index] - 1);
						// if mean time difference is less than 10, that's electronic noise, so we remove them all.
						if(dt_mean[index] < 10.0) {
//							printf("Skip cluster...\n");
							for(cluster_iAH_arr_cur[index] = 0; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].detectorID = 0;
							}
						}
						// otherwise, we remove them all except first and last
						else {
//							printf("Skip cluster...\n");
							for(cluster_iAH_arr_cur[index] = 1; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].detectorID = 0;
							}
						}
					}
					cluster_iAH_arr_size[index] = 0;
				}
				// current hit and previous hit are in same detector plane and in neighbor wires: 
				// we count how many hits we have in this case, until we find a hit in a different detector or in a wire that is not a neighbor to the previous hit.
				cluster_iAH_arr[index][cluster_iAH_arr_size[index]] = iAH[index];
				++cluster_iAH_arr_size[index];
			}
		}
	}
	//end of the hit loop

	// Hit reduction: 
	// store in "AllHits" containers only hits with non-zero detectorID and couting those with nAH_reduced
	for(iAH[index] = 0; iAH[index]<ic[index].nAH; ++iAH[index]) {
		if(ic[index].AllHits[iAH[index]].detectorID != 0) {
			ic[index].AllHits[nAH_reduced[index]] = ic[index].AllHits[iAH[index]];
			++nAH_reduced[index];

						
		}
	}

	// compute hits per detector
	int nEventHits = nAH_reduced[index];
	// reinitialize number of hits per detector
	for(auto iDetector = 1; iDetector <= nDetectors; ++iDetector) {
		ic[index].NHits[iDetector] = 0;
	}
	// loop on reduced hits and counting number of hits per detector
	for(auto iHit = 0; iHit < nEventHits; ++iHit) {
		auto detectorId = ic[index].AllHits[iHit].detectorID;
		if(detectorId != 0) {
			++ic[index].NHits[detectorId];
		}
	}


	ic[index].nAH = nAH_reduced[index];
	//ic[index].NHits[0] = NHits_reduced[index];
	


	//if(((ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])>0) || ((ic[index].NHits[7]+ic[index].NHits[8]+ic[index].NHits[9]+ic[index].NHits[10]+ic[index].NHits[11]+ic[index].NHits[12])>0) || ((ic[index].NHits[13]+ic[index].NHits[14]+ic[index].NHits[15]+ic[index].NHits[16]+ic[index].NHits[17]+ic[index].NHits[18])>0) || ((ic[index].NHits[19]+ic[index].NHits[20]+ic[index].NHits[21]+ic[index].NHits[22]+ic[index].NHits[23]+ic[index].NHits[24])>0) || ((ic[index].NHits[25]+ic[index].NHits[26]+ic[index].NHits[27]+ic[index].NHits[28]+ic[index].NHits[29]+ic[index].NHits[30])>0)){	
	//if(((ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])<270) || ((ic[index].NHits[7]+ic[index].NHits[8]+ic[index].NHits[9]+ic[index].NHits[10]+ic[index].NHits[11]+ic[index].NHits[12])>350) || ((ic[index].NHits[13]+ic[index].NHits[14]+ic[index].NHits[15]+ic[index].NHits[16]+ic[index].NHits[17]+ic[index].NHits[18])>170) || ((ic[index].NHits[19]+ic[index].NHits[20]+ic[index].NHits[21]+ic[index].NHits[22]+ic[index].NHits[23]+ic[index].NHits[24])>140) || ((ic[index].NHits[25]+ic[index].NHits[26]+ic[index].NHits[27]+ic[index].NHits[28]+ic[index].NHits[29]+ic[index].NHits[30])>140))

	//we do not accept the event unless there is at least one hit in the first DC

	/*
	if( (ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])<1){
		//printf("Event rejected...\n");
		}
		else {
			//counting total hit number, for all events < 6668? why? because she wanted just a subset!
			if( (ic[index].EventID)>10000 && (ic[index].EventID)<10050 ){//just look at a subset with something in it
				int totalDetectorHits = 0;
				for(int i = 1; i <= nDetectors; ++i) {
					totalDetectorHits += ic[index].NHits[i];
					//printf("%d ", ic[index].NHits[i]);
				}//printf("\n");
				
				//int nFirstRegionHits = 0;
				//for(int i = 1; i < 6; ++i) {
					//nFirstRegionHits += ic[index].NHits[i];
					//printf("nHits[%d] = %d\n", i, ic[index].NHits[i]);
				//}
			
				//printf("AllHits value : %d\n", (ic[index].NHits[0]));
				//printf("event : %d\n", (ic[index].EventID));
				//printf("all detector hit sum : %d; reduced AllHits value : %d\n", totalDetectorHits, (nAH_reduced[index]));
				//printf("sum of detectors : %d (%d)\n", totalDetectorHits, nFirstRegionHits);
				//}
			}
		    
	//		Process the accepted events (tracking) here.
			// where is the tracking though?
		}
	*/
}

// function to make the hit pairs in station;
// I assume it will only be called by the tracklet builder
// (not by the main function), so I can make it a "device" function. 
__device__ int make_hitpairs_in_station(gEvent* ic, thrust::pair<int, int>* hitpairs, int stID, int projID){
	   //can't see those from outside... so be it...
	   double spacingplane[28] = {0., 0.40, 0.40, 0.40, 0.40, 0.40, 0.40, 1.3, 1.3, 1.3, 1.2, 1.2, 1.2, 1.2, 1.2, 1.2, 4.0, 4.0, 7.0, 7.0, 8.0, 12.0, 12.0, 10.0, 3.0, 3.0, 3.0, 3.0};
	   double detsuperid[7][3] = {{2, 1, 3}, {5, 6, 4}, {8, 9, 7}, {11, 12, 10}, {14, 15, 13}, {25, 26, -1}, {24, 27, -1}}; 

	   int npairs = 0;
	   
	   // I think we assume that by default we want to know where we are
	   int index = threadIdx.x + blockIdx.x * blockDim.x;

	   //declaring arrays for the hit lists	   
	   int hitidx1[100]; 
	   int hitidx2[100];
	   int hitflag1[100]; 
	   int hitflag2[100];
	   for(int i = 0; i<100; i++){
	   	   hitidx1[i] = hitidx2[i] = 0;
		   hitflag1[i] = hitflag2[i] = 0;
	   }

	   //building the lists of hits for each detector plane
	   int detid1 = detsuperid[stID][projID]*2;
	   int detid2 = detsuperid[stID][projID]*2-1;
	   int superdetid = detsuperid[stID][projID];
	   int hitctr1 = 0, hitctr2 = 0;
	   for(int i = 0; i<ic[index].nAH; i++){
	   	  if(ic[index].AllHits[i].detectorID==detid1){
			hitidx1[hitctr1] = i;
			hitctr1++;
		  }
	   	  if(ic[index].AllHits[i].detectorID==detid2){
			hitidx2[hitctr2] = i;
			hitctr2++;
		  }
	   }

	   // pair the hits by position:
	   // if one hit on e.g. x and one hit on x' are closer than
	   // the "spacing" defined for the plane, then the hits can be paired together.
	   int idx1 = -1;
	   int idx2 = -1;
	   for(int i = 0; i<hitctr1; i++){
	   	   idx1++;
		   idx2 = -1;
	   	   for(int j = 0; j<hitctr2; j++){
		   	   idx2++;
			   if( abs(ic[index].AllHits[ hitidx1[idx1] ].pos - ic[index].AllHits[ hitidx2[idx2] ].pos) > spacingplane[superdetid] ){
			       continue;
			   }
			   	   
			   hitpairs[npairs] = thrust::make_pair(hitidx1[idx1], hitidx2[idx2]);
			   npairs++;
			   hitflag1[idx1] = 1;
			   hitflag2[idx2] = 1;
	   	   }
	   }
	   // here the hits that cannot be paired to another hit are paired to "nothing"
	   // (but they still have to be paired to be used in the trackletteing)
	   for(int i = 0; i<hitctr1; i++){
	   	   if(hitflag1[i]<1){
			hitpairs[npairs] = thrust::make_pair(hitidx1[i], -1);
			npairs++;
		   }
	   }
	   for(int i = 0; i<hitctr2; i++){
	   	   if(hitflag2[i]<1){
			hitpairs[npairs] = thrust::make_pair(hitidx2[i], -1);
			npairs++;
		   }
	   }
	   	   
	   return npairs;
}


// tracklet in station builder: 
__global__ void gkernel_TrackletinStation(gEvent* ic, gSW* oc, int stID, gPlane* planes) {
	// I think we assume that by default we want to know where we are
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	stID--;
	oc[index].EventID = ic[index].EventID;
	oc[index].nAH = ic[index].nAH;
	
	//if(10000<ic[index].EventID && ic[index].EventID<10050){
	//	for(int m = 0; m<30; m++){
	//		if(planes[m].u_win!=0)printf("index= %d, m = %d, u_win = %1.6f, costheta = %1.6f\n", index, m, planes[m].u_win, planes[m].costheta);
	//	}
	//}
	// loop on hits
	//if( (ic[index].EventID)>10000 && (ic[index].EventID)<10100 ){//just look at a subset with something in it
	//	printf("core idx %d, evt %d: reduced AllHits value : %d\n", (index), ic[index].EventID, (Nhits));
	//}
	// answer is yes, we still have the info from the previous function i.e. running after eR we still benefit from hit reduction;
	// was worth checking, just in case...

	//we don't need pairs of *HITS* necessarily, we just need pairs of indices...
	thrust::pair<int, int> hitpairs_x[100];
	thrust::pair<int, int> hitpairs_u[100];
	thrust::pair<int, int> hitpairs_v[100];

	int nx = make_hitpairs_in_station(ic, hitpairs_x, stID, 0);
	int nu = make_hitpairs_in_station(ic, hitpairs_u, stID, 1);
	int nv = make_hitpairs_in_station(ic, hitpairs_v, stID, 2);
	
	bool print = false;
	//if(ic[index].EventID==10042){
	//      print = true;
	//	printf("evt %d, nx = %d, nu = %d, nv = %d, uwin(plane %d) = %1.6f\n", ic[index].EventID, nx, nu, nv, stID*6, planes[stID*6].u_win);
	//}
	//one has to have at least one hit in x, u, v
	if(nx==0 || nu==0 || nv==0)return;
	
	int n_tkl = 0;
	
	//X-U combinations first
	for(int i = 0; i< nx; i++){
		double xpos = hitpairs_x[i].second>=0 ? 0.5*(ic[index].AllHits[ hitpairs_x[i].first ].pos+ic[index].AllHits[ hitpairs_x[i].second ].pos): ic[index].AllHits[ hitpairs_x[i].first ].pos;
		//u index for this station is simply stID*6
		double umin = xpos*planes[stID*6].costheta*planes[stID*6].u_win;
		double umax = umin+2*planes[stID*6].u_win;
		//if(print){
		//	printf("evt %d, xpos = %1.6f, umin = %1.6f, umax = %1.6f\n", ic[index].EventID, xpos, umin, umax);
		//	printf("x1 pos = %1.6f, x2 pos =%1.6f\n", ic[index].AllHits[ hitpairs_x[i].first ].pos, 
		//		hitpairs_x[i].second >=0 ? ic[index].AllHits[ hitpairs_x[i].second ].pos : -1000000);
		//}
		for(int j = 0; j< nu; j++){
			double upos = hitpairs_u[j].second>=0 ? 0.5*(ic[index].AllHits[ hitpairs_u[j].first ].pos+ic[index].AllHits[ hitpairs_u[j].second ].pos): ic[index].AllHits[ hitpairs_u[j].first ].pos;

			if(upos<umin || upos>umax)continue;
			//if(print)printf("evt %d, %1.6f <? upos = %1.6f <? %1.6f \n", ic[index].EventID, umin, upos, umax);
			
			double z_x = hitpairs_x[i].second>=0 ? planes[ ic[index].AllHits[ hitpairs_x[i].first ].detectorID ].z_mean : planes[ ic[index].AllHits[ hitpairs_x[i].first ].detectorID ].z;
			double z_u = hitpairs_u[j].second>=0 ? planes[ ic[index].AllHits[ hitpairs_u[j].first ].detectorID ].z_mean : planes[ ic[index].AllHits[ hitpairs_u[j].first ].detectorID ].z;
			//v index for this station is simply stID*6+4
			double z_v = planes[stID*6+4].z_mean;

			double v_win1 = planes[ ic[index].AllHits[ hitpairs_u[j].first ].detectorID ].v_win_fac1;
			double v_win2 = fabs(z_u+z_v-2*z_x)*planes[ stID*6+4 ].v_win_fac2;
			double v_win3 = fabs(z_v-z_u)*planes[ stID*6+4 ].v_win_fac3;
			double v_win = v_win1+v_win2+v_win3+2*planes[ ic[index].AllHits[ hitpairs_u[j].first ].detectorID ].spacing;

			double vmin = 2*xpos*planes[stID*6].costheta-upos-v_win;
			double vmax = vmin+2*v_win;

			for(int k = 0; k< nv; k++){
				double vpos = hitpairs_v[k].second>=0 ? 0.5*(ic[index].AllHits[ hitpairs_v[k].first ].pos+ic[index].AllHits[ hitpairs_v[k].second ].pos): ic[index].AllHits[ hitpairs_v[k].first ].pos;
				if(vpos<vmin || vpos>vmax)continue;
				int nhits_tkl = 0;
				oc[index].AllTracklets[n_tkl].stationID = stID;
				if(hitpairs_x[i].first>=0){
					oc[index].AllTracklets[n_tkl].hits[nhits_tkl]=ic[index].AllHits[ hitpairs_x[i].first ];
					oc[index].AllTracklets[n_tkl].nXHits++;
				}
				if(hitpairs_x[i].second>=0){
					oc[index].AllTracklets[n_tkl].hits[nhits_tkl]=ic[index].AllHits[ hitpairs_x[i].second ];
					oc[index].AllTracklets[n_tkl].nXHits++;
				}
				if(hitpairs_u[j].first>=0){
					oc[index].AllTracklets[n_tkl].hits[nhits_tkl]=ic[index].AllHits[ hitpairs_u[j].first ];
					oc[index].AllTracklets[n_tkl].nUHits++;
				}
				if(hitpairs_u[j].second>=0){
					oc[index].AllTracklets[n_tkl].hits[nhits_tkl]=ic[index].AllHits[ hitpairs_u[j].second ];
					oc[index].AllTracklets[n_tkl].nUHits++;
				}
				if(hitpairs_v[k].first>=0){
					oc[index].AllTracklets[n_tkl].hits[nhits_tkl]=ic[index].AllHits[ hitpairs_v[k].first ];
					oc[index].AllTracklets[n_tkl].nVHits++;
				}
				if(hitpairs_v[k].second>=0){
					oc[index].AllTracklets[n_tkl].hits[nhits_tkl]=ic[index].AllHits[ hitpairs_v[k].second ];
					oc[index].AllTracklets[n_tkl].nVHits++;
				}
				if(n_tkl>TrackletSizeMax)printf("evt %d: n_tkl = %d > %d\n", oc[index].EventID, n_tkl, TrackletSizeMax);
				n_tkl++;
			}
			
		}
	}
	//if(print)printf("evt %d number of tracklets %d\n", oc[index].EventID, n_tkl);
	oc[index].nTracklets = n_tkl;
}


// test code

#include <hiprand.h>

// // Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
// void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
// 	// Create a pseudo-random number generator
// 	hiprandGenerator_t prng;
// 	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

// 	// Set the seed for the random number generator using the system clock
// 	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

// 	// Fill the array with random numbers on the device
// 	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
// }

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const thrust::device_vector<float> &A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

// /**
//  * Online sample
//  * @see https://github.com/sol-prog/cuda_cublas_curand_thrust/blob/master/mmul_2.cu
//  */
// int main(int argc, char* argv[]) {
// 	// Allocate 3 arrays on CPU
// int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
 
// // for simplicity we are going to use square arrays
// nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
 
// thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A), d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);
 
// // Fill the arrays A and B on GPU with random numbers
// GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
// GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
 
// // Optionally we can print the data
// std::cout << "A =" << std::endl;
// print_matrix(d_A, nr_rows_A, nr_cols_A);
// std::cout << "B =" << std::endl;
// print_matrix(d_B, nr_rows_B, nr_cols_B);
 
// // Multiply A and B on GPU
// gpu_blas_mmul(thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]), thrust::raw_pointer_cast(&d_C[0]), nr_rows_A, nr_cols_A, nr_cols_B);
 
// //Print the result
// std::cout << "C =" << std::endl;
// print_matrix(d_C, nr_rows_C, nr_cols_C);
// }




int main(int argn, char * argv[]) {
	
	// initialization: declaration of SRaw event, opening file/tree, affecting rawEvent object to input tree
	// declaring array of gEvent;
	auto start = std::chrono::system_clock::now();
	clock_t cp1 = clock();

	TString inputFile;
	TString inputGeom;
	TString outputFile;
	inputFile = argv[1];
	inputGeom = argv[2];	
	outputFile = argv[3];

	//by default we should use e1039 
	bool e906data = false;
	if(argn>=4)e906data = atoi(argv[4]);

	cout<<"Running "<<argv[0]<<endl;
	cout<<"Loading "<<argv[1]<<endl;
	cout<<"with geometry: "<<argv[2]<<endl;
	cout<<"Writing "<<argv[3]<<endl;
	
	//Get basic geometry here:
	double u_factor[5] = {5., 5., 5., 15., 15.};
	gPlane plane[nChamberPlanes+nHodoPlanes+nPropPlanes];
	ifstream in_geom(inputGeom.Data());
  	string buffer;
	int ipl, nelem;
	double z, spacing, xoffset, scalex, x0, costheta, scaley, y0, sintheta, deltaW_;
 	while ( getline(in_geom, buffer) ) {
    	      if (buffer[0] == '#') continue;
	      std::istringstream iss;
	      iss.str(buffer);
	      iss >> ipl >> z >> nelem >> spacing >> xoffset >> scalex >> x0 >> costheta >> scaley >> y0 >> sintheta;
	      plane[ipl-1].z = z;
	      plane[ipl-1].nelem = nelem;
	      plane[ipl-1].spacing = spacing;
	      plane[ipl-1].xoffset = xoffset;
	      plane[ipl-1].scalex = scalex;
	      plane[ipl-1].x0 = x0;
	      plane[ipl-1].costheta = costheta;
	      plane[ipl-1].scaley = scaley;
	      plane[ipl-1].y0 = y0;
	      plane[ipl-1].sintheta = sintheta;
	      if(ipl>nChamberPlanes+nHodoPlanes){
		for(int k = 0; k<9; k++){
			iss >> deltaW_;
			plane[ipl-1].deltaW_[k] = deltaW_;
		}
	      }else{
		iss >> deltaW_;
		plane[ipl-1].deltaW_[0] = deltaW_;
	      }
	      ipl++;
	}
	
	for(int i = 0; i<5; i++){
		int u_idx = i*6; 
		int x_idx = i*6+2;
		for(int j = 0; j<6; j++){
			int idx = i*6+j;
			plane[idx].z_mean = j%2==0 ? 0.5*(plane[idx].z+plane[idx-1].z):0.5*(plane[idx].z+plane[idx+1].z);
			
			plane[idx].v_win_fac1 = plane[idx].spacing*2*plane[u_idx].costheta;
			plane[idx].v_win_fac2 = plane[u_idx].costheta*TX_MAX;
			plane[idx].v_win_fac3 = plane[u_idx].sintheta*TY_MAX;
		}
		
		for(int j = 0; j<6; j++){
			int idx = i*6+j;
			plane[idx].u_win = fabs(0.5*plane[u_idx].scaley*plane[u_idx].sintheta) + TX_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].costheta) + TY_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].sintheta) + 2.*plane[i].spacing + u_factor[i];
		}
		//cout << i*6 << " " << plane[i*6].u_win << endl;
	}
	cout << "Geometry file read out" << endl;
	
	std::unordered_map<int, double> map_elemPosition[nChamberPlanes+nHodoPlanes+nPropPlanes+1];
	for(int i = 0; i < nChamberPlanes; ++i){
		cout << plane[i].nelem << endl;
      		for(int j = 0; j < plane[i].nelem; ++j){
          		double pos = (j - (plane[i].nelem+1.)/2.)*plane[i].spacing + plane[i].xoffset + plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].deltaW_[0];
          		map_elemPosition[i].insert(posType(j, pos));
			
		}
	}
	for(int i = nChamberPlanes; i<nChamberPlanes+nHodoPlanes; ++i){
		cout << plane[i].nelem << endl;
	      	for(int j = 0; j < plane[i].nelem; ++j){
          		double pos = plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].xoffset + (j - (plane[i].nelem+1)/2.)*plane[i].spacing + plane[i].deltaW_[0];
          		map_elemPosition[i].insert(posType(j, pos));
		}
	}
	for(int i = nChamberPlanes+nHodoPlanes; i<nChamberPlanes+nHodoPlanes+nPropPlanes; ++i){
		cout << plane[i].nelem << endl;
	      	for(int j = 0; j < plane[i].nelem; ++j){
          		int moduleID = 8 - int((j - 1)/8);
			//cout << moduleID << endl;
             		double pos = plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].xoffset + (j - (plane[i].nelem+1)/2.)*plane[i].spacing + plane[i].deltaW_[moduleID];
          		map_elemPosition[i].insert(posType(j, pos));
		}
		
	}
	
	TFile* dataFile = new TFile(inputFile.Data(), "READ");
	TTree* dataTree = (TTree *)dataFile->Get("save");
	SRawEvent* rawEvent = new SRawEvent();
	SQEvent* Event = new SQEvent();
	if(e906data){
		dataTree->SetBranchAddress("rawEvent", &rawEvent);
	}else{
		dataTree->SetBranchAddress("SQEvent", &Event);
		//default option
	}
	int nEvtMax = dataTree->GetEntries();
	static gEvent host_gEvent[EstnEvtMax];

	cout << "unfolding events" << endl;
	// loop on event: get RawEvent information and load it into gEvent
	for(int i = 0; i < nEvtMax; ++i) {
		dataTree->GetEntry(i);
//		cout<<"Converting "<<i<<"/"<<nEvtMax<<endl;
		if(e906data){
			host_gEvent[i].RunID = rawEvent->fRunID;
			host_gEvent[i].EventID = rawEvent->fEventID;
			host_gEvent[i].SpillID = rawEvent->fSpillID;
			host_gEvent[i].TriggerBits = rawEvent->fTriggerBits;
			host_gEvent[i].TargetPos = rawEvent->fTargetPos;
			host_gEvent[i].TurnID = rawEvent->fTurnID;
			host_gEvent[i].RFID = rawEvent->fRFID;
			for(int j=0; j<33; j++) {
				host_gEvent[i].Intensity[j] = rawEvent->fIntensity[j];
			}
			host_gEvent[i].TriggerEmu = rawEvent->fTriggerEmu;
			for(int k=0; k<4; k++) {
				host_gEvent[i].NRoads[k] = rawEvent->fNRoads[k];
			}
			for(int l=0; l<(nChamberPlanes+nHodoPlanes+nPropPlanes+1); l++) {
				host_gEvent[i].NHits[l] = rawEvent->fNHits[l];
			}
			host_gEvent[i].nAH = rawEvent->fAllHits.size();
			host_gEvent[i].nTH = rawEvent->fTriggerHits.size();
			for(int m=0; m<rawEvent->fAllHits.size(); m++) {
				host_gEvent[i].AllHits[m].index=(rawEvent->fAllHits[m]).index;
				host_gEvent[i].AllHits[m].detectorID=(rawEvent->fAllHits[m]).detectorID;
				host_gEvent[i].AllHits[m].elementID=(rawEvent->fAllHits[m]).elementID;
				host_gEvent[i].AllHits[m].tdcTime=(rawEvent->fAllHits[m]).tdcTime;
				host_gEvent[i].AllHits[m].driftDistance=(rawEvent->fAllHits[m]).driftDistance;
				host_gEvent[i].AllHits[m].pos=map_elemPosition[(rawEvent->fAllHits[m]).detectorID-1][(rawEvent->fAllHits[m]).elementID];
				host_gEvent[i].AllHits[m].flag=(rawEvent->fAllHits[m]).flag;
			}
			for(int n=0; n<rawEvent->fTriggerHits.size(); n++) {
				host_gEvent[i].TriggerHits[n].index=(rawEvent->fTriggerHits[n]).index;
				host_gEvent[i].TriggerHits[n].detectorID=(rawEvent->fTriggerHits[n]).detectorID;
				host_gEvent[i].TriggerHits[n].elementID=(rawEvent->fTriggerHits[n]).elementID;
				host_gEvent[i].TriggerHits[n].tdcTime=(rawEvent->fTriggerHits[n]).tdcTime;
				host_gEvent[i].TriggerHits[n].driftDistance=(rawEvent->fTriggerHits[n]).driftDistance;
				host_gEvent[i].TriggerHits[n].pos=map_elemPosition[(rawEvent->fAllHits[n]).detectorID-1][(rawEvent->fAllHits[n]).elementID];
				host_gEvent[i].TriggerHits[n].flag=(rawEvent->fTriggerHits[n]).flag;
			}
			// printouts for test
			//if(10000<rawEvent->fEventID&&rawEvent->fEventID<10050){
			//	printf("%d:\n ", rawEvent->fEventID);
			//	for(int l = 1; l<=nChamberPlanes; l++){
			//		printf("%d ", rawEvent->fNHits[l]);
			//	}printf("; %d\n", rawEvent->fAllHits.size());
			//	for(int m = 0; m<=50; m++){
			//		printf("%d, %1.3f;", (rawEvent->fAllHits[m]).detectorID, (rawEvent->fAllHits[m]).pos);
			//	}printf("\n");
			//}
		}else{
			//Default option: e1039
			//Adding this code will not be obvious without a file
		}
	}
	cout << "loaded events" << endl;


//If the decoded has NOT been sorted...
//	for(int i = 0; i < nEvtMax; ++i) {
//		thrust::stable_sort(host_gEvent[i].AllHits, host_gEvent[i].AllHits+host_gEvent[i].nAH, lessthan());
//	}


	// evaluate the total size of the gEvent array (and the SW array) for memory allocation 
	// (the memory cannot be dynamically allocated) 
	size_t NBytesAllEvent = EstnEvtMax * sizeof(gEvent);
	size_t NBytesAllSearchWindow = EstnEvtMax * sizeof(gSW);
	size_t NBytesAllPlanes =  nDetectors * sizeof(gPlane);

	gEvent *host_output_eR = (gEvent*)malloc(NBytesAllEvent);
	gSW *host_output_TKL = (gSW*)malloc(NBytesAllSearchWindow);
	
	// declaring gEvent objects for the device (GPU) to use.
	gEvent *device_gEvent;
	// gEvent *device_output_eR;
	// gEvent *device_input_TKL;
	gSW *device_output_TKL;
	gPlane *device_gPlane;
	
	// copy of data from host to device: evaluate operation time 
	clock_t cp2 = clock();
	
	// printDeviceStatus();
	// Allocating memory for GPU (pointer to allocated device ); check for errors in the process; stops the program if issues encountered
	gpuErrchk( hipMalloc((void**)&device_gEvent, NBytesAllEvent));
	//gpuErrchk( hipMalloc((void**)&device_input_TKL, NBytesAllEvent));
	gpuErrchk( hipMalloc((void**)&device_output_TKL, NBytesAllSearchWindow));
	//allocating the memory for the planes, just in case...
	gpuErrchk( hipMalloc((void**)&device_gPlane, NBytesAllPlanes));

	// hipMemcpy(dst, src, count, kind): copies data between host and device:
	// dst: destination memory address; src: source memory address; count: size in bytes; kind: type of transfer
	// hipMalloc((void**)&device_output_eR, sizeofoutput_eR);
	gpuErrchk( hipMemcpy(device_gEvent, host_gEvent, NBytesAllEvent, hipMemcpyHostToDevice));
	// gpuErrchk( hipMemcpy(device_output_TKL, host_output_TKL, NBytesAllEvent, hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpy(device_gPlane, plane, NBytesAllPlanes, hipMemcpyHostToDevice));
	// hipMemcpy(device_output_eR, host_output, sizeofoutput_eR, hipMemcpyHostToDevice);
	
	// now data is transfered in the device: kernel function for event reconstruction called;
	// note that the function call is made requesting a number of blocks and a number of threads per block
	// in practice we have as many threads total as number of events; 
	auto start_er = std::chrono::system_clock::now();
	gkernel_eR<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent);
	auto end_er = std::chrono::system_clock::now();
	
	// check status of device and synchronize;
	size_t nEvents = EstnEvtMax;
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// copy result of event reconstruction from device_gEvent to device_input_TKL
	// this input_tkl should be the information that the device uses to reconstruct the tracklets
	// gpuErrchk( hipMemcpy(device_input_TKL, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToDevice));

	// shouldn't this function actually be called? should it be the function that puts together tracklets? and then call the fitting???
	// gkernel_TKL<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_input_TKL, device_output_TKL);

	//for(int m = 0; m<30; m++){
	//	if(plane[m].u_win!=0)printf("plane, m = %d, u_win = %1.6f, costheta = %1.6f\n", m, plane[m].u_win, plane[m].costheta);
	//	if(device_gPlane[m].u_win!=0)printf("device_gplane, m = %d, u_win = %1.6f, costheta = %1.6f\n", m, device_gPlane[m].u_win, device_gPlane[m].costheta);
	//}

	auto start_tkl = std::chrono::system_clock::now();
	// I first want to see if indeed we can reuse the "gEvent" pointer
	int stID = 3;// to make explicit that we are requiring station 3
	gkernel_TrackletinStation<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent, device_output_TKL, stID, device_gPlane);
	auto end_tkl = std::chrono::system_clock::now();
	cout << endl;
	// check status of device and synchronize again;
	
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// data transfer from device to host
	gpuErrchk( hipMemcpy(host_output_eR, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToHost));

	gpuErrchk( hipMemcpy(host_output_TKL, device_output_TKL, NBytesAllSearchWindow, hipMemcpyDeviceToHost));

	// thrust objects: C++ template library based on STL
	// convert raw pointer device_gEvent to device_vector
	// TODO: just don't use raw pointers to begin with
	thrust::device_ptr<gEvent> d_p_events(device_gEvent);
	thrust::device_vector<gEvent> d_events(d_p_events, d_p_events + nEvents);
    	
	std::vector<gEvent> h_events(nEvents);
	std::copy(d_events.begin(), d_events.end(), h_events.begin());

	thrust::device_vector<float> d_hit_pos(nEvents);
	// std::vector<float> h_hit_pos;

	// copy hit pos from event vector to dedicated hit pos vector
	// TODO: do this on the GPU instead (possibly using zip_iterator)
	// for (auto j = h_events.begin(); j < h_events.begin() + 100; ++j) {
	// 	// cout << "e " << j->EventID << endl;
	// 	for (auto i = 0; i < EstnAHMax; ++i) {
	// 		// float pos = static_cast<gEvent>(*j).AllHits[i].pos;
	// 		float pos = static_cast<gEvent>(*j).AllHits[i].driftDistance;
	// 		if (abs(pos) > Epsilon) {
	// 			// h_hit_pos.push_back(pos);
	// 			d_hit_pos.push_back(pos);
	// 			// cout  << " " << pos << endl;
	// 		}
	// 	}
	// }
	// thrust::copy(h_hit_pos.begin(), h_hit_pos.end(), d_hit_pos.begin());


	// thrust::transform(device_gEvent, device_gEvent + nEvents, d_hit_pos.begin(), get_first_event_hit_pos());
	// cout << "First event positions (10 / " << d_hit_pos.size() << "):";
	// thrust::copy(d_hit_pos.begin(), d_hit_pos.begin()+10, std::ostream_iterator<int>(std::cout, ", "));
	// cout <<  endl;

	// int NCheck = 10;
	// for (int i = 0; i < NCheck; ++i) {
	// 	gEvent& evt = (host_output_eR)[i];
	// 	gSW& sw = (host_output_TKL)[i];
	// 	cout <<  i << ". " << evt.EventID <<  ". " <<  evt.nAH << ", " << sw.EventID << ", " << sw.nAH << endl;
	// }
	
	
	/*
	// ###############################################################
	// Gpufit
	// ###############################################################
	
	thrust::device_vector< REAL > d_parameters(2);
	
	//data is array of xz-positions of v,v',x,x',u,u' planes of each tracklet
	vector< REAL > _data_tkl_x {-2.48f,-2.50f, -0.824f, -0.826f, -0.473f,-0.474f};
	thrust::device_vector<REAL> d_tkl_x(_data_tkl_x.size());
	thrust::copy(_data_tkl_x.begin(), _data_tkl_x.end(), d_tkl_x.begin());


	// true parameters fo xz view
	std::vector< REAL > true_parameters_x { 150, 0.15f }; // offset, slope
	thrust::copy(true_parameters_x.begin(), true_parameters_x.end(), d_parameters.begin());

	// linear_regression_example(d_hit_pos.size(), d_hit_pos.data().get());
	// calling linear regression with 6 points... 
	// it looks like we're fitting a single tracklet. 
	// Where is the part where we're getting all tracklet candidates and fit them? 
	// what about the part where we are fitting a full track?
	linear_regression_example(d_tkl_x.size(), d_tkl_x.data().get(), d_parameters);

	//data is array of yz-positions of v,v',x,x',y,y' planes of each tracklet

	vector< REAL > _data_tkl_y {-0.761f, -0.764f, -0.067f, -0.069f, -0.742f, -0.75f};
	thrust::device_vector<REAL> d_tkl_y(_data_tkl_y.size());
	thrust::copy(_data_tkl_y.begin(), _data_tkl_y.end(), d_tkl_y.begin());


	// true parameters fo yz view
	std::vector< REAL > true_parameters_y { 150, 0.15f }; // offset, slope
	thrust::copy(true_parameters_y.begin(), true_parameters_y.end(), d_parameters.begin());

	linear_regression_example(d_tkl_y.size(), d_tkl_y.data().get(), d_parameters);
	*/


	hipMemcpy(host_gEvent, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToHost);
	// hipMemcpy(host_output, device_output_eR, sizeofoutput_eR, hipMemcpyDeviceToHost);
	// hipFree(device_gEvent);
	// // hipFree(device_output_eR);
	// hipFree(device_input_TKL);
	// hipFree(device_output_TKL);

	//auto end_kernel = std::chrono::system_clock::now();
	clock_t cp3 = clock();

	delete rawEvent;

	TFile* outFile = new TFile(outputFile.Data(), "RECREATE");
	/*
	//TTree* ORoutput_tree = new TTree("OR_out", "OR_out");
	ORoutput_tree* output = new ORoutput_tree();
	for(int i = 0; i < nEvtMax; ++i) {
		output->Clear();
		for(int k = 1; k<=nDetectors; k++ )output->fNhitsReduced[k] = host_output_eR[i].NHits[k];
		//output->Write();
	}
	output->Write();
	*/

	// printouts for test
	//for(int i = 1; i < nEvtMax; ++i) {
	//	//if(10000<host_gEvent[i].EventID && host_gEvent[i].EventID<10050){
	//	if(host_gEvent[i].EventID==10042){
	//		printf("%d:\n ", host_gEvent[i].EventID);
	//		for(int j = 0; j<=nChamberPlanes; j++){
	//			printf("%d ", host_gEvent[i].NHits[j]);
	//		}printf("; %d\n", host_gEvent[i].nAH);
	//		for(int j = 0; j<=host_gEvent[i].nAH; j++){
	//			if(13<=host_gEvent[i].AllHits[j].detectorID&&host_gEvent[i].AllHits[j].detectorID<=18)printf("%d, %1.3f; ", host_gEvent[i].AllHits[j].detectorID, host_gEvent[i].AllHits[j].pos);
	//		}
	//		printf("\n");
	//	}
	//}
		
	//for(int i = 0; i < host_gEvent[0].nAH; ++i) {
		//cout<<"D0_1st_wire:" << (host_gEvent[0].NHits[1])<<endl;
		//cout<<"output: "<<(host_gEvent[0].nAH)<<endl;
		//cout<<"output: "<<(device_output_eR)<<endl;
		//cout<<"output: "<<(sizeof(int))<<endl;
		//cout<<"size: "<<i<<endl;
	//}
	// printing the time required for all operations
	clock_t cp4 = clock();
	auto end = std::chrono::system_clock::now();

	//double cpu_secs = double(cp4-cp3+cp2-cp1) / CLOCKS_PER_SEC;
	double cpu_secs = double(cp2-cp1) / CLOCKS_PER_SEC;
	auto gpu_er = end_er - start_er;
	auto gpu_tkl = end_tkl - start_tkl;
	auto overall = end - start;
	cout<<"Read/prepare events: "<<cpu_secs<<endl;
	cout<<"event reducing: "<<(gpu_er.count()/1000000000.0)<<endl;
	cout<<"trackletting: "<<(gpu_tkl.count()/1000000000.0)<<endl;
	cout<<"Total time: "<<(overall.count()/1000000000.0)<<endl;

	return 0;
}
