#include "hip/hip_runtime.h"
#include "gpufit.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <string>
#include <ctime>
#include <chrono>

// CUDA runtime
// #include <hip/hip_runtime.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

// #include <hipblaslt.h>


#include <TObject.h>
#include <TROOT.h>
#include <TFile.h>
#include <TTree.h>
#include <TRandom.h>
#include <TMatrixD.h>
#include <TLorentzVector.h>
#include <TClonesArray.h>
#include <TStopwatch.h>
#include <TTimeStamp.h>
#include <TString.h>
//#include "LoadInput.h"
#include "OROutput.h"

#include "SRawEvent.h"
#include "GeomSvc.h"
#include "KalmanFastTracking.h"

#include "SQEvent_v1.h"
#include "SQHit_v1.h"
#include "SQHitVector_v1.h"

#define nChamberPlanes 30
#define nHodoPlanes 16
#define nPropPlanes 8
#define nDetectors (nChamberPlanes+nHodoPlanes+nPropPlanes)
#define Epsilon 0.00001f

#define triggerBit(n) (1 << (n))
#define hitFlagBit(n) (1 << (n))

using namespace std;

const int EstnEvtMax = 10240;
const int THREADS_PER_BLOCK = 512;
int BLOCKS_NUM = EstnEvtMax/THREADS_PER_BLOCK;
const int EstnAHMax = 5000;
const int EstnTHMax = 200;
const int ClusterSizeMax = 100;

// function to check GPU status
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// looping on all GPUs to check their status
void printDeviceStatus() {
	int nDevices;

	
	gpuErrchk( hipGetDeviceCount(&nDevices) );
	for (int i = 0; i < nDevices; i++) {
	  hipDeviceProp_t prop;
	  hipGetDeviceProperties(&prop, i);
	  printf("Device Number: %d\n", i);
	  printf("  Device name: %s\n", prop.name);
	  printf("  Memory Clock Rate (KHz): %d\n",
			 prop.memoryClockRate);
	  printf("  Memory Bus Width (bits): %d\n",
			 prop.memoryBusWidth);
	  printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}
}





//clone of LoadEvent::Hit:
class gHit {
	public:
	int index; // global hit index in the hit array
	short detectorID; // ID of the detector: one ID for each DC wire plane (30 total), hodoscope plane (16 total), proportional tube plane (8 total).
	short elementID; // ID of the element in the detector: wire/slat/tube number
	float tdcTime; // raw TDC time from the DAQ 
	float driftDistance; // calculated drift distance from RT profile (supplied in database) IF tdcTime between tmin and tmax defined for detector; 
	float pos; // position in the projection of the detector (e.g. X in a X plane, etc)
	short flag; // 1: in time; 2: hodo mask; 3: trigger mask
};

class gEvent {
	public:
	int RunID; // Run Number
	int EventID; // Event number
	int SpillID; // Spill number
	int TriggerBits; // hash of the trigger bits: 0-4: MATRIX1-5; 5-9: NIM1-5;
	short TargetPos; // target position: proxy for target ID?
	int TurnID; // => related to beam intensity
	int RFID; // => related to beam intensity
	int Intensity[33]; //  16 before, one onset, and 16 after
	short TriggerEmu; // 1 if MC event
	short NRoads[4]; // 0, positive top; 1, positive bottom; 2, negative top; 3, negative bottom
	int NHits[nDetectors+1]; // number of hits in each detector plane
	int nAH; // size of AllHits
	int nTH; // size of TriggerHits
	gHit AllHits[EstnAHMax]; // array of all hits
	gHit TriggerHits[EstnTHMax]; // array of trigger hits
};

// SW = SearchWindows?
class gSW {
public:
	int EventID;
	int nAH;
};

// Hit comparison
struct lessthan {
	__host__ __device__ bool operator()(const gHit& lhs, const gHit& rhs)
	{
	//returns true if :
	// hit1.detID<hit2.detID;  
		if(lhs.detectorID < rhs.detectorID)
		{
			return true;
		}
		else if(lhs.detectorID > rhs.detectorID)
		{
			return false;
		}
	//hit1.detID=hit2.detID & hit1.elID<hit2.elID;
		if(lhs.elementID < rhs.elementID)
		{
			return true;
		}
		else if(lhs.elementID > rhs.elementID)
		{
			return false;
		}
	//hit1.detID=hit2.detID & hit1.elID=hit2.elID & hit1.time>hit2.time;
		if(lhs.tdcTime > rhs.tdcTime)
		{
			return true;
		}
		else
		{
			return false;
		}
	}
};


// position of first hit 
struct get_first_event_hit_pos
{
  __host__ __device__
  float operator()(gEvent& evt)
  {
    return evt.AllHits[0].pos;
  }
};

// Linear regression: fit of tracks
void linear_regression_example(int n_points_per_fit, REAL *device_input, thrust::device_vector<REAL> &d_parameters)
{
	// number of fits, fit points and parameters
	size_t const n_fits = 1;
	size_t const n_model_parameters = 2;


	// custom z positions, stored in user info
	// NOTE: this is the way to initialize a device_vector with many constant values
	vector< REAL > user_info_values {-130.43f, -131.437f, -116.782f, -115.712f, -128.114f, -129.111f };
	thrust::device_vector<REAL> d_user_info(user_info_values.size());
	thrust::copy(user_info_values.begin(), user_info_values.end(), d_user_info.begin());

	// size of user info in bytes
	size_t const user_info_size = d_user_info.size() * sizeof(REAL); 

	// // initial parameters (randomized)
	// std::vector< REAL > initial_parameters(n_fits * n_model_parameters);
	// for (size_t i = 0; i != n_fits; i++)
	// {
	// 	// random offset
	// 	initial_parameters[i * n_model_parameters + 0] = true_parameters[0] * (0.8f + 0.4f * uniform_dist(rng));
	// 	// random slope
	// 	initial_parameters[i * n_model_parameters + 1] = true_parameters[0] * (0.8f + 0.4f * uniform_dist(rng));
	// }

	// // generate data
	// std::vector< REAL > data(n_points_per_fit * n_fits);
	// for (size_t i = 0; i != data.size(); i++)
	// {

	// 	size_t j = i / n_points_per_fit; // the fit
	// 	size_t k = i % n_points_per_fit; // the position within a fit

	// 	REAL x = user_info[k];
	// 	REAL y = true_parameters[0] + x * true_parameters[1];
	// 	data[i] = y;



	// }

	// tolerance
	REAL const tolerance = 0.001f;

	// maximum number of iterations
	int const max_number_iterations = 20;

	// estimator ID
	int const estimator_id = LSE;

	// model ID
	int const model_id = LINEAR_1D;

	// parameters to fit (all of them)
	std::vector< int > parameters_to_fit(n_model_parameters, 1);

	
	thrust::device_vector< int > d_states(n_fits);
	thrust::device_vector< REAL > d_chi_square(n_fits);
	thrust::device_vector< int > d_number_iterations(n_fits);

	//call to gpufit (C interface)
	// can be found in https://github.com/gpufit/Gpufit/blob/master/Gpufit/gpufit.cpp
	// parameters:
	// size_t n_fits,
    	// size_t n_points,
	// float * gpu_data,
	// float * gpu_weights,
    	// int model_id,
    	// float tolerance,
    	// int max_n_iterations,
    	// int * parameters_to_fit,
    	// int estimator_id,
	// size_t user_info_size,
    	// char * gpu_user_info,
    	// float * gpu_fit_parameters,
    	// int * gpu_output_states,
    	// float * gpu_output_chi_squares,
    	// int * gpu_output_n_iterations
	// size_t n_fits,

	int const status = gpufit_cuda_interface
       (
        	n_fits,
        	n_points_per_fit,
            device_input,
            0,
            model_id,
            // initial_parameters.data(),
            tolerance,
            max_number_iterations,
            parameters_to_fit.data(),
			// true_parameters.data(),
            estimator_id,
            user_info_size,
            reinterpret_cast< char * >( thrust::raw_pointer_cast(d_user_info.data()) ),
            thrust::raw_pointer_cast(d_parameters.data()),
            thrust::raw_pointer_cast(d_states.data()),
            thrust::raw_pointer_cast(d_chi_square.data()),
            thrust::raw_pointer_cast(d_number_iterations.data())
        );

		

	// check status
	if (status != ReturnState::OK)
	{
		throw std::runtime_error(gpufit_get_last_error());
	}

	// get fit states
	std::vector< int > output_states_histogram(5, 0);
	for (auto it = d_states.begin(); it != d_states.end(); ++it)
	{
		output_states_histogram[*it]++;
	}

	std::cout << "ratio converged              " << (REAL) output_states_histogram[0] / n_fits << "\n";
	std::cout << "ratio max iteration exceeded " << (REAL) output_states_histogram[1] / n_fits << "\n";
	std::cout << "ratio singular hessian       " << (REAL) output_states_histogram[2] / n_fits << "\n";
	std::cout << "ratio neg curvature MLE      " << (REAL) output_states_histogram[3] / n_fits << "\n";
	std::cout << "ratio gpu not read           " << (REAL) output_states_histogram[4] / n_fits << "\n";

	// compute mean fitted parameters for converged fits
	std::vector< REAL > output_parameters_mean(n_model_parameters, 0);
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			// add offset
			output_parameters_mean[0] += d_parameters[i * n_model_parameters + 0];
			// add slope
			output_parameters_mean[1] += d_parameters[i * n_model_parameters + 1];
		}
	}
	output_parameters_mean[0] /= output_states_histogram[0];
	output_parameters_mean[1] /= output_states_histogram[0];

	// compute std of fitted parameters for converged fits
	std::vector< REAL > output_parameters_std(n_model_parameters, 0);
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			// add squared deviation for offset
			output_parameters_std[0] += (d_parameters[i * n_model_parameters + 0] - output_parameters_mean[0]) * (d_parameters[i * n_model_parameters + 0] - output_parameters_mean[0]);
			// add squared deviation for slope
			output_parameters_std[1] += (d_parameters[i * n_model_parameters + 1] - output_parameters_mean[1]) * (d_parameters[i * n_model_parameters + 1] - output_parameters_mean[1]);
		}
	}
	// divide and take square root
	output_parameters_std[0] = sqrt(output_parameters_std[0] / output_states_histogram[0]);
	output_parameters_std[1] = sqrt(output_parameters_std[1] / output_states_histogram[0]);

	// print mean and std
	std::cout << "offset  true " << d_parameters[0] << " mean " << output_parameters_mean[0] << " std " << output_parameters_std[0] << "\n";
	std::cout << "slope   true " << d_parameters[1] << " mean " << output_parameters_mean[1] << " std " << output_parameters_std[1] << "\n";

	// compute mean chi-square for those converged
	REAL  output_chi_square_mean = 0;
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			output_chi_square_mean += d_chi_square[i];
		}
	}
	output_chi_square_mean /= static_cast<REAL>(output_states_histogram[0]);
	std::cout << "mean chi square " << output_chi_square_mean << "\n";

	// compute mean number of iterations for those converged
	REAL  output_number_iterations_mean = 0;
	for (size_t i = 0; i != n_fits; i++)
	{
		if (d_states[i] == FitState::CONVERGED)
		{
			output_number_iterations_mean += static_cast<REAL>(d_number_iterations[i]);
		}
	}

	// normalize
	output_number_iterations_mean /= static_cast<REAL>(output_states_histogram[0]);
	std::cout << "mean number of iterations " << output_number_iterations_mean << "\n";
}


// kernel functions: 
// CUDA C++ extends C++ by allowing the programmer to define C++ functions, called kernels, that, when called, 
// are executed N times in parallel by N different CUDA threads, as opposed to only once like regular C++ functions. 
// I guess TKL is for tracklet selection? 
__global__ void gkernel_TKL(gEvent* ic, gSW* sw) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	sw[index].nAH = ic[index].nAH;
	sw[index].EventID = ic[index].EventID;
}

// event reducer: 
__global__ void gkernel_eR(gEvent* ic) {
	//printf("Running the kernel function...\n");
	// retrieve global thread index
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	double w_max[EstnEvtMax]; // max drift distance of the hit furthest from the cluster avg position // current average position of cluster * 0.9
	
	double w_min[EstnEvtMax]; // max drift distance of the hit closest to the cluster avg position // current average position of cluster * 0.4
	double dt_mean[EstnEvtMax]; // tbd
	int cluster_iAH_arr_cur[EstnEvtMax]; // current cluster array
	int cluster_iAH_arr_size[EstnEvtMax]; // cluster size i.e number of hits in cluster
	static int cluster_iAH_arr[EstnEvtMax][ClusterSizeMax]; // global cluster array 
	int uniqueID[EstnEvtMax]; // hit unique element ID
	int uniqueID_curr[EstnEvtMax]; // current hit unique element ID
	double tdcTime_curr[EstnEvtMax]; // current hit TDC time
	int iAH[EstnEvtMax]; // hit index 
	int nAH_reduced[EstnEvtMax]; // number of hits after hit quality filtering
	// int nHitsPerDetector[nDetectors+1];
	

	// initialization of array size
	cluster_iAH_arr_size[index] = 0;
	nAH_reduced[index] = 0;
		
	// event reducing/hit filtering
	for(iAH[index] = 0; iAH[index]<ic[index].nAH; ++iAH[index]) {
		// if hit not good, set its detID to 0 and continue;
		if((ic[index].AllHits[iAH[index]].flag & hitFlagBit(1)) == 0) {
//			printf("Skip out-of-time...\n");
			ic[index].AllHits[iAH[index]].detectorID = 0;
			continue;
		}
		// hits in DCs or Prop tubes
		if(ic[index].AllHits[iAH[index]].detectorID < 31 || ic[index].AllHits[iAH[index]].detectorID > 46) {
			// evaluate "unique ID"
			uniqueID[index] = ic[index].AllHits[iAH[index]].detectorID*1000 + ic[index].AllHits[iAH[index]].elementID;
			// compare with current unique element ID; if different, update the unique element ID and time info 
			if(uniqueID[index] != uniqueID_curr[index]) {
				uniqueID_curr[index] = uniqueID[index];
				tdcTime_curr[index] = ic[index].AllHits[iAH[index]].tdcTime;
			}
			// if next hit and current hit belong to the same element: if detID>36 => prop tubes (reminder that hodoscpes are out of the picture in this scope), 
			// we're suppose to have one signal (a second signal would be after-pulsing)
			// if time difference between new hit and current hit is less than 80ns for DCs, it's also considered after-pulsing
			else {
				if(ic[index].AllHits[iAH[index]].detectorID > 36 || ((ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] >= 0.0) && (ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] < 80.0)) || ((ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] <= 0.0) && (ic[index].AllHits[iAH[index]].tdcTime - tdcTime_curr[index] > -80.0))) {
//					printf("Skip after-pulse...\n");
					ic[index].AllHits[iAH[index]].detectorID = 0;
					continue;
				}
				else {
					tdcTime_curr[index] = ic[index].AllHits[iAH[index]].tdcTime;
				}
			}
		}
		// declustering of hits in DCs (from CPU code, I understand this one better)
		// if there are hits in the same plane and hitting two neighboring wires, they both give redundant information: 
		if(ic[index].AllHits[iAH[index]].detectorID <= nChamberPlanes) {
//			printf("%d\n", cluster_iAH_arr_size[index]);
//			printf("Decluster...\n");
			if(cluster_iAH_arr_size[index] == ClusterSizeMax) {
//				printf("Oversized cluster...\n");
			}
			// if array size is zero, start storing the hit in the array
			if(cluster_iAH_arr_size[index] == 0) {
				cluster_iAH_arr[index][0] = iAH[index];
				++cluster_iAH_arr_size[index];
			}
			// otherwise
			else {
				// current hit and previous hit are *not* in same detector plane OR next hit and current hit are *not* in neighbors cells
				// we "declusterize" i.e. we remove the hit/hits which information is redundant with other hits and/or useless
				if((ic[index].AllHits[iAH[index]].detectorID != ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID) || (ic[index].AllHits[iAH[index]].elementID - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].elementID > 1)) {
					// if 2 hits in cluster, evaluate w_max and w_min; drift distance has to be < w_min for one of the hits, while it has to be < w_max for the other hit 
					if(cluster_iAH_arr_size[index] == 2) {
						w_max[index] = 0.9*0.5*(ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].pos - ic[index].AllHits[cluster_iAH_arr[index][0]].pos);
						w_min[index] = 4.0/9.0*w_max[index];
						if((ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > w_max[index] && ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance > w_min[index]) || (ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > w_min[index] && ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance > w_max[index])) {
							//eliminating the existing hit with the lagest drift distance
							if(ic[index].AllHits[cluster_iAH_arr[index][0]].driftDistance > ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].driftDistance) {
//								printf("Skip cluster...\n");
								ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID = 0;
							}
							else {
//								printf("Skip cluster...\n");
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID = 0;
							}
						}
						// if the time difference is less than 8 ns for detectors 19 to 24 (which btw are DC3p) we remove both
						else if((((ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) >= 0.0 && (ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) < 8.0) || ((ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) <= 0.0 && (ic[index].AllHits[cluster_iAH_arr[index][0]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].tdcTime) > -8.0)) && (ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID >= 19 && ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID <= 24)) {
//							printf("Skip cluster...\n");
							ic[index].AllHits[cluster_iAH_arr[index][0]].detectorID = 0;
							ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_size[index]-1]].detectorID = 0;
						}
					}
					// if 3 hits or more in cluster: we essentially discard them all;
					if(cluster_iAH_arr_size[index] >= 3) {
						// evaluate the mean time difference;
						dt_mean[index] = 0.0;
						for(cluster_iAH_arr_cur[index] = 1; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
							dt_mean[index] += ((ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime) > 0.0 ? (ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime) : (ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]-1]].tdcTime - ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].tdcTime));
						}
						dt_mean[index] = dt_mean[index]/(cluster_iAH_arr_size[index] - 1);
						// if mean time difference is less than 10, that's electronic noise, so we remove them all.
						if(dt_mean[index] < 10.0) {
//							printf("Skip cluster...\n");
							for(cluster_iAH_arr_cur[index] = 0; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].detectorID = 0;
							}
						}
						// otherwise, we remove them all except first and last
						else {
//							printf("Skip cluster...\n");
							for(cluster_iAH_arr_cur[index] = 1; cluster_iAH_arr_cur[index] < cluster_iAH_arr_size[index]; ++cluster_iAH_arr_cur[index]) {
								ic[index].AllHits[cluster_iAH_arr[index][cluster_iAH_arr_cur[index]]].detectorID = 0;
							}
						}
					}
					cluster_iAH_arr_size[index] = 0;
				}
				// current hit and previous hit are in same detector plane and in neighbor wires: 
				// we count how many hits we have in this case, until we find a hit in a different detector or in a wire that is not a neighbor to the previous hit.
				cluster_iAH_arr[index][cluster_iAH_arr_size[index]] = iAH[index];
				++cluster_iAH_arr_size[index];
			}
		}
	}
	//end of the hit loop

	// Hit reduction: 
	// store in "AllHits" containers only hits with non-zero detectorID and couting those with nAH_reduced
	for(iAH[index] = 0; iAH[index]<ic[index].nAH; ++iAH[index]) {
		if(ic[index].AllHits[iAH[index]].detectorID != 0) {
			ic[index].AllHits[nAH_reduced[index]] = ic[index].AllHits[iAH[index]];
			++nAH_reduced[index];

						
		}
	}

	// compute hits per detector
	int nEventHits = nAH_reduced[index];
	// reinitialize number of hits per detector
	for(auto iDetector = 1; iDetector <= nDetectors; ++iDetector) {
		ic[index].NHits[iDetector] = 0;
	}
	// loop on reduced hits and counting number of hits per detector
	for(auto iHit = 0; iHit < nEventHits; ++iHit) {
		auto detectorId = ic[index].AllHits[iHit].detectorID;
		if(detectorId != 0) {
			++ic[index].NHits[detectorId];
		}
	}


	ic[index].nAH = nAH_reduced[index];
	//ic[index].NHits[0] = NHits_reduced[index];
	


	//if(((ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])>0) || ((ic[index].NHits[7]+ic[index].NHits[8]+ic[index].NHits[9]+ic[index].NHits[10]+ic[index].NHits[11]+ic[index].NHits[12])>0) || ((ic[index].NHits[13]+ic[index].NHits[14]+ic[index].NHits[15]+ic[index].NHits[16]+ic[index].NHits[17]+ic[index].NHits[18])>0) || ((ic[index].NHits[19]+ic[index].NHits[20]+ic[index].NHits[21]+ic[index].NHits[22]+ic[index].NHits[23]+ic[index].NHits[24])>0) || ((ic[index].NHits[25]+ic[index].NHits[26]+ic[index].NHits[27]+ic[index].NHits[28]+ic[index].NHits[29]+ic[index].NHits[30])>0)){	
	//if(((ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])<270) || ((ic[index].NHits[7]+ic[index].NHits[8]+ic[index].NHits[9]+ic[index].NHits[10]+ic[index].NHits[11]+ic[index].NHits[12])>350) || ((ic[index].NHits[13]+ic[index].NHits[14]+ic[index].NHits[15]+ic[index].NHits[16]+ic[index].NHits[17]+ic[index].NHits[18])>170) || ((ic[index].NHits[19]+ic[index].NHits[20]+ic[index].NHits[21]+ic[index].NHits[22]+ic[index].NHits[23]+ic[index].NHits[24])>140) || ((ic[index].NHits[25]+ic[index].NHits[26]+ic[index].NHits[27]+ic[index].NHits[28]+ic[index].NHits[29]+ic[index].NHits[30])>140))

	//we do not accept the event unless there is at least one hit in the first DC

	if( (ic[index].NHits[1]+ic[index].NHits[2]+ic[index].NHits[3]+ic[index].NHits[4]+ic[index].NHits[5]+ic[index].NHits[6])<1){
		//printf("Event rejected...\n");
		}
		else {
			//counting total hit number, for all events < 6668? why? because she wanted just a subset!
			if( (ic[index].EventID)>10000 && (ic[index].EventID)<10100 ){//just look at a subset with something in it
				int totalDetectorHits = 0;
				for(int i = 1; i <= nDetectors; ++i) {
					totalDetectorHits += ic[index].NHits[i];
				}
	
				int nFirstRegionHits = 0;
				for(int i = 1; i < 6; ++i) {
					nFirstRegionHits += ic[index].NHits[i];
					printf("nHits[%d] = %d\n", i, ic[index].NHits[i]);
			}
			
				// printf("AllHits value : %d\n", (ic[index].NHits[0]));
				//printf("event : %d\n", (ic[index].EventID));
				//printf("reduced AllHits value : %d\n", (nAH_reduced[index]));
				//printf("sum of detectors : %d (%d)\n", totalDetectorHits, nFirstRegionHits);
				//}
			}
		    
	//		Process the accepted events (tracking) here.
			// where is the tracking though?
		}

}


// tracklet in station builder: 
__global__ void gkernel_TrackletinStation(gEvent* ic, int stID) {
	// I think we assume that by default we want to know where we are
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	// loop on hits
	int Nhits = ic[index].nAH;
	//if( (ic[index].EventID)>10000 && (ic[index].EventID)<10100 ){//just look at a subset with something in it
	//	printf("core idx %d, evt %d: reduced AllHits value : %d\n", (index), ic[index].EventID, (Nhits));
	//}
	// answer is yes, we still have the info from the previous function i.e. running this function after running eR still offers 
	
}



// test code

#include <hiprand.h>

// // Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
// void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
// 	// Create a pseudo-random number generator
// 	hiprandGenerator_t prng;
// 	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

// 	// Set the seed for the random number generator using the system clock
// 	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

// 	// Fill the array with random numbers on the device
// 	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
// }

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const thrust::device_vector<float> &A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

// /**
//  * Online sample
//  * @see https://github.com/sol-prog/cuda_cublas_curand_thrust/blob/master/mmul_2.cu
//  */
// int main(int argc, char* argv[]) {
// 	// Allocate 3 arrays on CPU
// int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
 
// // for simplicity we are going to use square arrays
// nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
 
// thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A), d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);
 
// // Fill the arrays A and B on GPU with random numbers
// GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
// GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
 
// // Optionally we can print the data
// std::cout << "A =" << std::endl;
// print_matrix(d_A, nr_rows_A, nr_cols_A);
// std::cout << "B =" << std::endl;
// print_matrix(d_B, nr_rows_B, nr_cols_B);
 
// // Multiply A and B on GPU
// gpu_blas_mmul(thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]), thrust::raw_pointer_cast(&d_C[0]), nr_rows_A, nr_cols_A, nr_cols_B);
 
// //Print the result
// std::cout << "C =" << std::endl;
// print_matrix(d_C, nr_rows_C, nr_cols_C);
// }




int main(int argn, char * argv[]) {
	
	// initialization: declaration of SRaw event, opening file/tree, affecting rawEvent object to input tree
	// declaring array of gEvent;
	auto start = std::chrono::system_clock::now();
	clock_t cp1 = clock();

	TString inputFile;
	TString outputFile;
	inputFile = argv[1];
	inputGeom = argv[2];	
	outputFile = argv[3];

	cout << argn << endl;

	//by default we should use e1039 
	bool e906data = false;
	if(argn>4)e906data = atoi(argv[4]);

	cout<<"Running "<<argv[0]<<endl;
	cout<<"Loading "<<argv[1]<<endl;
	cout<<"with geometry: "<<argv[2]<<endl;
	cout<<"Writing "<<argv[3]<<endl;
	
	//Get basic geometry here:
	double u_factor[5] = {5., 5., 5., 15., 15.};
	gPlane plane[nChamberPlanes+nHodoPlanes+nPropPlanes];
	ifstream in_geom(inputGeom.Data());
  	string buffer;
	int ipl, nelem;
	double z, spacing, xoffset, scalex, x0, costheta, scaley, y0, sintheta, deltaW_;
 	while ( getline(in_geom, buffer) ) {
    	      if (buffer[0] == '#') continue;
	      std::istringstream iss;
	      iss.str(buffer);
	      iss >> ipl >> z >> nelem >> spacing >> xoffset >> scalex >> x0 >> costheta >> scaley >> y0 >> sintheta;
	      plane[ipl-1].z = z;
	      plane[ipl-1].nelem = nelem;
	      plane[ipl-1].spacing = spacing;
	      plane[ipl-1].xoffset = xoffset;
	      plane[ipl-1].scalex = scalex;
	      plane[ipl-1].x0 = x0;
	      plane[ipl-1].costheta = costheta;
	      plane[ipl-1].scaley = scaley;
	      plane[ipl-1].y0 = y0;
	      plane[ipl-1].sintheta = sintheta;
	      if(ipl>nChamberPlanes+nHodoPlanes){
		for(int k = 0; k<9; k++){
			iss >> deltaW_;
			plane[ipl-1].deltaW_[k] = deltaW_;
		}
	      }else{
		iss >> deltaW_;
		plane[ipl-1].deltaW_[0] = deltaW_;
	      }
	      ipl++;
	}
	
	for(int i = 0; i<5; i++){
		int u_idx = i*6; 
		int x_idx = i*6+2;
		for(int j = 0; j<6; j++){
			int idx = i*6+j;
			plane[idx].z_mean = j%2==0 ? 0.5*(plane[idx].z+plane[idx-1].z):0.5*(plane[idx].z+plane[idx+1].z);
			
			plane[idx].v_win_fac1 = plane[idx].spacing*2*plane[u_idx].costheta;
			plane[idx].v_win_fac2 = plane[u_idx].costheta*TX_MAX;
			plane[idx].v_win_fac3 = plane[u_idx].sintheta*TY_MAX;
		}
		
		for(int j = 0; j<6; j++){
			int idx = i*6+j;
			plane[idx].u_win = fabs(0.5*plane[u_idx].scaley*plane[u_idx].sintheta) + TX_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].costheta) + TY_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].sintheta) + 2.*plane[i].spacing + u_factor[i];
		}
		//cout << i*6 << " " << plane[i*6].u_win << endl;
	}
	cout << "Geometry file read out" << endl;
	
	std::unordered_map<int, double> map_elemPosition[nChamberPlanes+nHodoPlanes+nPropPlanes+1];
	for(int i = 0; i < nChamberPlanes; ++i){
		//cout << plane[i].nelem << endl;
      		for(int j = 1; j <= plane[i].nelem; ++j){
          		double pos = (j - (plane[i].nelem+1.)/2.)*plane[i].spacing + plane[i].xoffset + plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].deltaW_[0];
          		map_elemPosition[i].insert(posType(j, pos));
			
		}
	}
	for(int i = nChamberPlanes; i<nChamberPlanes+nHodoPlanes; ++i){
		//cout << plane[i].nelem << endl;
	      	for(int j = 1; j <= plane[i].nelem; ++j){
          		double pos = plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].xoffset + (j - (plane[i].nelem+1)/2.)*plane[i].spacing + plane[i].deltaW_[0];
          		map_elemPosition[i].insert(posType(j, pos));
		}
	}
	for(int i = nChamberPlanes+nHodoPlanes; i<nChamberPlanes+nHodoPlanes+nPropPlanes; ++i){
		//cout << plane[i].nelem << endl;
	      	for(int j = 1; j <= plane[i].nelem; ++j){
          		int moduleID = 8 - int((j - 1)/8);
			//cout << moduleID << endl;
             		double pos = plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].xoffset + (j - (plane[i].nelem+1)/2.)*plane[i].spacing + plane[i].deltaW_[moduleID];
          		map_elemPosition[i].insert(posType(j, pos));
		}
		
	}
	
	TFile* dataFile = new TFile(inputFile.Data(), "READ");
	TTree* dataTree = 0;// = (TTree *)dataFile->Get("save");
	SRawEvent* rawEvent = new SRawEvent();
	//SQEvent_v1* event = new SQEvent_v1();
	//SQHitVector_v1* hitvec = new SQHitVector_v1();
	
	Int_t     _run_id = 0;
 	Int_t     _spill_id = 0;
	Int_t     _event_id = 0;
   	UShort_t  _trigger;
	Int_t     _qie_presums[4];
	Int_t     _qie_turn_id;
	Int_t     _qie_rf_id;
	Int_t     _qie_rf_inte[33];
	
	std::vector<SQHit*> hit_vec;
	
	
	if(e906data){
		dataTree = (TTree *)dataFile->Get("save");
		dataTree->SetBranchAddress("rawEvent", &rawEvent);
	}else{
		//default option: e1039
		dataTree = (TTree *)dataFile->Get("T");
		dataTree->SetMakeClass(1); //this is necessary to get the tree to read the branch correctly
		dataTree->SetBranchStatus("*", 0);// this speeds up the loop on events
		
		dataTree->SetBranchStatus("DST.SQEvent._run_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._spill_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._event_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._trigger", 1);
		dataTree->SetBranchStatus("DST.SQEvent._qie_presums[4]", 1);
		dataTree->SetBranchStatus("DST.SQEvent._qie_turn_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._qie_rf_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._qie_rf_inte[33]", 1);
		
		dataTree->SetBranchAddress("DST.SQEvent._run_id", &_run_id);
		dataTree->SetBranchAddress("DST.SQEvent._spill_id", &_spill_id);
		dataTree->SetBranchAddress("DST.SQEvent._event_id", &_event_id);
		dataTree->SetBranchAddress("DST.SQEvent._trigger", &_trigger);
		dataTree->SetBranchAddress("DST.SQEvent._qie_presums[4]", &_qie_presums);
		dataTree->SetBranchAddress("DST.SQEvent._qie_turn_id", &_qie_turn_id);
		dataTree->SetBranchAddress("DST.SQEvent._qie_rf_id", &_qie_rf_id);
		dataTree->SetBranchAddress("DST.SQEvent._qie_rf_inte[33]", &_qie_rf_inte[33]);
		
		dataTree->SetBranchStatus("DST.SQHitVector._vector", 1);
		dataTree->SetBranchAddress("DST.SQHitVector._vector", &hit_vec);
	}
	int nEvtMax = dataTree->GetEntries();
	static gEvent host_gEvent[EstnEvtMax];

	cout << "unfolding " << nEvtMax <<" events" << endl;
	
	// loop on event: get RawEvent information and load it into gEvent
	for(int i = 0; i < nEvtMax; ++i) {
		if(i%1000==0)cout << i << "/" << nEvtMax <<  endl;
		dataTree->GetEntry(i);
		//cout<<"Converting "<<i<<"/"<<nEvtMax<<endl;
		if(e906data){
			host_gEvent[i].RunID = rawEvent->fRunID;
			host_gEvent[i].EventID = rawEvent->fEventID;
			host_gEvent[i].SpillID = rawEvent->fSpillID;
			host_gEvent[i].TriggerBits = rawEvent->fTriggerBits;
			host_gEvent[i].TargetPos = rawEvent->fTargetPos;
			host_gEvent[i].TurnID = rawEvent->fTurnID;
			host_gEvent[i].RFID = rawEvent->fRFID;
			for(int j=0; j<33; j++) {
				host_gEvent[i].Intensity[j] = rawEvent->fIntensity[j];
			}
			host_gEvent[i].TriggerEmu = rawEvent->fTriggerEmu;
			for(int k=0; k<4; k++) {
				host_gEvent[i].NRoads[k] = rawEvent->fNRoads[k];
			}
			for(int l=0; l<(nChamberPlanes+nHodoPlanes+nPropPlanes+1); l++) {
				host_gEvent[i].NHits[l] = rawEvent->fNHits[l];
			}
			host_gEvent[i].nAH = rawEvent->fAllHits.size();
			host_gEvent[i].nTH = rawEvent->fTriggerHits.size();
			for(int m=0; m<rawEvent->fAllHits.size(); m++) {
				host_gEvent[i].AllHits[m].index=(rawEvent->fAllHits[m]).index;
				host_gEvent[i].AllHits[m].detectorID=(rawEvent->fAllHits[m]).detectorID;
				host_gEvent[i].AllHits[m].elementID=(rawEvent->fAllHits[m]).elementID;
				host_gEvent[i].AllHits[m].tdcTime=(rawEvent->fAllHits[m]).tdcTime;
				host_gEvent[i].AllHits[m].driftDistance=(rawEvent->fAllHits[m]).driftDistance;
				host_gEvent[i].AllHits[m].pos=map_elemPosition[(rawEvent->fAllHits[m]).detectorID][(rawEvent->fAllHits[m]).elementID];
				host_gEvent[i].AllHits[m].flag=(rawEvent->fAllHits[m]).flag;
			}
			for(int n=0; n<rawEvent->fTriggerHits.size(); n++) {
				host_gEvent[i].TriggerHits[n].index=(rawEvent->fTriggerHits[n]).index;
				host_gEvent[i].TriggerHits[n].detectorID=(rawEvent->fTriggerHits[n]).detectorID;
				host_gEvent[i].TriggerHits[n].elementID=(rawEvent->fTriggerHits[n]).elementID;
				host_gEvent[i].TriggerHits[n].tdcTime=(rawEvent->fTriggerHits[n]).tdcTime;
				host_gEvent[i].TriggerHits[n].driftDistance=(rawEvent->fTriggerHits[n]).driftDistance;
				host_gEvent[i].TriggerHits[n].pos=map_elemPosition[(rawEvent->fAllHits[n]).detectorID][(rawEvent->fAllHits[n]).elementID];
				host_gEvent[i].TriggerHits[n].flag=(rawEvent->fTriggerHits[n]).flag;
			}
			// printouts for test
			//if(10000<rawEvent->fEventID&&rawEvent->fEventID<10050){
			//	printf("%d:\n ", rawEvent->fEventID);
			//	for(int l = 1; l<=nChamberPlanes; l++){
			//		printf("%d ", rawEvent->fNHits[l]);
			//	}printf("; %d\n", rawEvent->fAllHits.size());
			//	for(int m = 0; m<=50; m++){
			//		printf("%d, %1.3f;", (rawEvent->fAllHits[m]).detectorID, (rawEvent->fAllHits[m]).pos);
			//	}printf("\n");
			//}
		}else{
			//Default option: e1039
			//if(i%1000==0){
			//	cout << i << " " << _event_id << " " << _trigger << " " << &hit_vec << " " << hit_vec.size() << endl;
			//	dataTree->Show(i);
			//}
			host_gEvent[i].RunID = _run_id;
			host_gEvent[i].SpillID = _spill_id;
			host_gEvent[i].EventID = _event_id;
			host_gEvent[i].TriggerBits = _trigger;
			for(int k = 0; k<4; k++)host_gEvent[i].NRoads[k] = _qie_presums[k];
			host_gEvent[i].TurnID = _qie_turn_id;
			host_gEvent[i].RFID = _qie_rf_id;
			for(int k = 0; k<33; k++)host_gEvent[i].Intensity[k] = _qie_rf_inte[k];
			
			int ntrighits = 0;
			host_gEvent[i].nAH = hit_vec.size();
			for(int m = 0; m<hit_vec.size(); m++){
				host_gEvent[i].AllHits[m].index=hit_vec[m]->get_hit_id();
				host_gEvent[i].AllHits[m].detectorID=hit_vec[m]->get_detector_id();
				host_gEvent[i].AllHits[m].elementID=hit_vec[m]->get_element_id();
				host_gEvent[i].AllHits[m].tdcTime=hit_vec[m]->get_tdc_time();
				host_gEvent[i].AllHits[m].driftDistance=hit_vec[m]->get_drift_distance();
				host_gEvent[i].AllHits[m].pos=hit_vec[m]->get_pos();
				if(hit_vec[m]->is_trigger_mask()){
					ntrighits++;
					host_gEvent[i].TriggerHits[m].index=hit_vec[m]->get_hit_id();
					host_gEvent[i].TriggerHits[m].detectorID=hit_vec[m]->get_detector_id();
					host_gEvent[i].TriggerHits[m].elementID=hit_vec[m]->get_element_id();
					host_gEvent[i].TriggerHits[m].tdcTime=hit_vec[m]->get_tdc_time();
					host_gEvent[i].TriggerHits[m].driftDistance=hit_vec[m]->get_drift_distance();
					host_gEvent[i].TriggerHits[m].pos=hit_vec[m]->get_pos();
				}
			}
			host_gEvent[i].nTH = ntrighits;
		}
	}
	cout << "loaded events" << endl;
	
//If the decoded has NOT been sorted...
//	for(int i = 0; i < nEvtMax; ++i) {
//		thrust::stable_sort(host_gEvent[i].AllHits, host_gEvent[i].AllHits+host_gEvent[i].nAH, lessthan());
//	}


	// evaluate the total size of the gEvent array (and the SW array) for memory allocation 
	// (the memory cannot be dynamically allocated) 
	size_t NBytesAllEvent = EstnEvtMax * sizeof(gEvent);
	size_t NBytesAllSearchWindow = EstnEvtMax * sizeof(gSW);

	gEvent *host_output_eR = (gEvent*)malloc(NBytesAllEvent);
	gSW * host_output_TKL = (gSW*)malloc(NBytesAllSearchWindow);

	// declaring gEvent objects for the device (GPU) to use.
	gEvent *device_gEvent;
	// gEvent *device_output_eR;
	gEvent *device_input_TKL;
	gSW *device_output_TKL;
	
	
	// copy of data from host to device: evaluate operation time 
	clock_t cp2 = clock();
	auto start_kernel = std::chrono::system_clock::now();

	// printDeviceStatus();
	// Allocating memory for GPU (pointer to allocated device ); check for errors in the process; stops the program if issues encountered
	gpuErrchk( hipMalloc((void**)&device_gEvent, NBytesAllEvent));
	gpuErrchk( hipMalloc((void**)&device_input_TKL, NBytesAllEvent));
	gpuErrchk( hipMalloc((void**)&device_output_TKL, NBytesAllSearchWindow));

	// hipMemcpy(dst, src, count, kind): copies data between host and device:
	// dst: destination memory address; src: source memory address; count: size in bytes; kind: type of transfer
	// hipMalloc((void**)&device_output_eR, sizeofoutput_eR);
	gpuErrchk( hipMemcpy(device_gEvent, host_gEvent, NBytesAllEvent, hipMemcpyHostToDevice));
	// hipMemcpy(device_output_eR, host_output, sizeofoutput_eR, hipMemcpyHostToDevice);
	auto end_kernel = std::chrono::system_clock::now();
	
	// now data is transfered in the device: kernel function for event reconstruction called;
	// note that the function call is made requesting a number of blocks and a number of threads per block
	// in practice we have as many threads total as number of events; 
	//auto start_kernel = std::chrono::system_clock::now();
	gkernel_eR<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent);
	//auto end_kernel = std::chrono::system_clock::now();
	
	// check status of device and synchronize;
	size_t nEvents = EstnEvtMax;
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// copy result of event reconstruction from device_gEvent to device_input_TKL
	// this input_tkl should be the information that the device uses to reconstruct the tracklets
	gpuErrchk( hipMemcpy(device_input_TKL, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToDevice));

	// shouldn't this function actually be called? should it be the function that puts together tracklets? and then call the fitting???
	// gkernel_TKL<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_input_TKL, device_output_TKL);

	// I first want to see if indeed we can reuse the "gEvent" pointer
	int stID = 3;// to make explicit that we are requiring station 3
	gkernel_TrackletinStation<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent, stID);

	// check status of device and synchronize again;
	
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// data transfer from device to host
	gpuErrchk( hipMemcpy(host_output_eR, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy(host_output_TKL, device_output_TKL, NBytesAllSearchWindow, hipMemcpyDeviceToHost));


	// thrust objects: C++ template library based on STL
	// convert raw pointer device_gEvent to device_vector
	// TODO: just don't use raw pointers to begin with
    thrust::device_ptr<gEvent> d_p_events(device_gEvent);
    thrust::device_vector<gEvent> d_events(d_p_events, d_p_events + nEvents);
	std::vector<gEvent> h_events(nEvents);
	std::copy(d_events.begin(), d_events.end(), h_events.begin());

	thrust::device_vector<float> d_hit_pos(nEvents);
	// std::vector<float> h_hit_pos;

	// copy hit pos from event vector to dedicated hit pos vector
	// TODO: do this on the GPU instead (possibly using zip_iterator)
	// for (auto j = h_events.begin(); j < h_events.begin() + 100; ++j) {
	// 	// cout << "e " << j->EventID << endl;
	// 	for (auto i = 0; i < EstnAHMax; ++i) {
	// 		// float pos = static_cast<gEvent>(*j).AllHits[i].pos;
	// 		float pos = static_cast<gEvent>(*j).AllHits[i].driftDistance;
	// 		if (abs(pos) > Epsilon) {
	// 			// h_hit_pos.push_back(pos);
	// 			d_hit_pos.push_back(pos);
	// 			// cout  << " " << pos << endl;
	// 		}
	// 	}
	// }
	// thrust::copy(h_hit_pos.begin(), h_hit_pos.end(), d_hit_pos.begin());


	// thrust::transform(device_gEvent, device_gEvent + nEvents, d_hit_pos.begin(), get_first_event_hit_pos());
	// cout << "First event positions (10 / " << d_hit_pos.size() << "):";
	// thrust::copy(d_hit_pos.begin(), d_hit_pos.begin()+10, std::ostream_iterator<int>(std::cout, ", "));
	// cout <<  endl;

	// int NCheck = 10;
	// for (int i = 0; i < NCheck; ++i) {
	// 	gEvent& evt = (host_output_eR)[i];
	// 	gSW& sw = (host_output_TKL)[i];
	// 	cout <<  i << ". " << evt.EventID <<  ". " <<  evt.nAH << ", " << sw.EventID << ", " << sw.nAH << endl;
	// }
	
	
	
	// ###############################################################
	// Gpufit
	// ###############################################################

	thrust::device_vector< REAL > d_parameters(2);
	
	//data is array of xz-positions of v,v',x,x',u,u' planes of each tracklet
	vector< REAL > _data_tkl_x {-2.48f,-2.50f, -0.824f, -0.826f, -0.473f,-0.474f};
	thrust::device_vector<REAL> d_tkl_x(_data_tkl_x.size());
	thrust::copy(_data_tkl_x.begin(), _data_tkl_x.end(), d_tkl_x.begin());


	// true parameters fo xz view
	std::vector< REAL > true_parameters_x { 150, 0.15f }; // offset, slope
	thrust::copy(true_parameters_x.begin(), true_parameters_x.end(), d_parameters.begin());

	// linear_regression_example(d_hit_pos.size(), d_hit_pos.data().get());
	// calling linear regression with 6 points... 
	// it looks like we're fitting a single tracklet. 
	// Where is the part where we're getting all tracklet candidates and fit them? 
	// what about the part where we are fitting a full track?
	linear_regression_example(d_tkl_x.size(), d_tkl_x.data().get(), d_parameters);

	//data is array of yz-positions of v,v',x,x',y,y' planes of each tracklet

	vector< REAL > _data_tkl_y {-0.761f, -0.764f, -0.067f, -0.069f, -0.742f, -0.75f};
	thrust::device_vector<REAL> d_tkl_y(_data_tkl_y.size());
	thrust::copy(_data_tkl_y.begin(), _data_tkl_y.end(), d_tkl_y.begin());


	// true parameters fo yz view
	std::vector< REAL > true_parameters_y { 150, 0.15f }; // offset, slope
	thrust::copy(true_parameters_y.begin(), true_parameters_y.end(), d_parameters.begin());

	linear_regression_example(d_tkl_y.size(), d_tkl_y.data().get(), d_parameters);



	// hipMemcpy(host_gEvent, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToHost);
	// hipMemcpy(host_output, device_output_eR, sizeofoutput_eR, hipMemcpyDeviceToHost);
	// hipFree(device_gEvent);
	// // hipFree(device_output_eR);
	// hipFree(device_input_TKL);
	// hipFree(device_output_TKL);

	//auto end_kernel = std::chrono::system_clock::now();
	clock_t cp3 = clock();

	delete rawEvent;

	//cout<<"output: "<<(device_output_eR)<<endl
	
	//for(int i = 0; i < host_gEvent[0].nAH; ++i) {
		  //cout<<"D0_1st_wire:" << (host_gEvent[0].NHits[1])<<endl;
		//cout<<"output: "<<(host_gEvent[0].nAH)<<endl;
		//cout<<"output: "<<(device_output_eR)<<endl;
		//cout<<"output: "<<(sizeof(int))<<endl;
		//cout<<"size: "<<i<<endl;
	//}
	// printing the time required for all operations
	clock_t cp4 = clock();
	auto end = std::chrono::system_clock::now();

	//double cpu_secs = double(cp4-cp3+cp2-cp1) / CLOCKS_PER_SEC;
	double cpu_secs = double(cp2-cp1) / CLOCKS_PER_SEC;
	auto gpu_ns = end_kernel - start_kernel;
	auto overall = end - start;
	cout<<"Read/prepare events: "<<cpu_secs<<endl;
	cout<<"eR "<<(gpu_ns.count()/1000000000.0)<<endl;
	cout<<"Total time: "<<(overall.count()/1000000000.0)<<endl;

	return 0;
}
