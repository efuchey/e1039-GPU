#include "hip/hip_runtime.h"
#include "gpufit.h"
#include "interface.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <fstream>
#include <string>
#include <ctime>
#include <chrono>

// CUDA runtime
// #include <hip/hip_runtime.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

// #include <hipblaslt.h>

#include <TObject.h>
#include <TROOT.h>
#include <TFile.h>
#include <TTree.h>
#include <TRandom.h>
#include <TMatrixD.h>
#include <TLorentzVector.h>
#include <TClonesArray.h>
#include <TStopwatch.h>
#include <TTimeStamp.h>
#include <TString.h>
//#include "LoadInput.h"
#include "OROutput.h"
#include "reconstruction_kernels.cuh"

#ifdef E1039
#include "SQEvent_v1.h"
#include "SQHit_v1.h"
#include "SQHitVector_v1.h"
#endif

// function to check GPU status
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// looping on all GPUs to check their status
void printDeviceStatus() {
	int nDevices;

	
	gpuErrchk( hipGetDeviceCount(&nDevices) );
	for (int i = 0; i < nDevices; i++) {
		 hipDeviceProp_t prop;
		 hipGetDeviceProperties(&prop, i);
		 printf("Device Number: %d\n", i);
		 printf("  Device name: %s\n", prop.name);
		 printf("  Memory Clock Rate (KHz): %d\n",
		 	prop.memoryClockRate);
		 printf("  Memory Bus Width (bits): %d\n",
		 	prop.memoryBusWidth);
		 printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		 	2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}
}


// Hit comparison
struct lessthan {
	__host__ __device__ bool operator()(const gHit& lhs, const gHit& rhs)
	{
	//returns true if :
	// hit1.detID<hit2.detID;  
		if(lhs.detectorID < rhs.detectorID)
		{
			return true;
		}
		else if(lhs.detectorID > rhs.detectorID)
		{
			return false;
		}
	//hit1.detID=hit2.detID & hit1.elID<hit2.elID;
		if(lhs.elementID < rhs.elementID)
		{
			return true;
		}
		else if(lhs.elementID > rhs.elementID)
		{
			return false;
		}
	//hit1.detID=hit2.detID & hit1.elID=hit2.elID & hit1.time>hit2.time;
		if(lhs.tdcTime > rhs.tdcTime)
		{
			return true;
		}
		else
		{
			return false;
		}
	}
};




int main(int argn, char * argv[]) {
	
	// initialization: declaration of SRaw event, opening file/tree, affecting rawEvent object to input tree
	// declaring array of gEvent;
	auto start = std::chrono::system_clock::now();
	auto cp1 = std::chrono::system_clock::now();

	TString inputFile;
	TString inputGeom;
	TString outputFile;
	inputFile = argv[1];
	inputGeom = argv[2];	
	outputFile = argv[3];

	//by default we should use e1039 
	bool e906data = true;
	//if(argn>4)e906data = atoi(argv[4]);
	
	cout<<"Running "<<argv[0]<<endl;
	cout<<"Loading "<<argv[1]<<endl;
	cout<<"with geometry: "<<argv[2]<<endl;
	cout<<"Writing "<<argv[3]<<endl;
	
	//Get basic geometry here:
	double u_factor[5] = {5., 5., 5., 15., 15.};
	gPlane plane[nChamberPlanes+nHodoPlanes+nPropPlanes+1];

	ifstream in_geom(inputGeom.Data());
  	string buffer;
	int ipl, nelem;
	float z, cellwidth, spacing, xoffset, scalex, x0, x1, x2, costheta, scaley, y0, y1, y2, sintheta, resolution, deltaW_;
	float p1x, p1y, p1z, deltapx, deltapy, deltapz, dp1x, dp1y, dp1z;
 	while ( getline(in_geom, buffer) ) {
    	      if (buffer[0] == '#') continue;
	      std::istringstream iss;
	      iss.str(buffer);
	      iss >> ipl >> z >> nelem >> cellwidth >> spacing >> xoffset >> scalex >> x0 >> x1 >> x2 >> costheta >> scaley >> y0 >> y1 >> y2 >> sintheta >> resolution >> p1x >> p1y >> p1z >> deltapx >> deltapy >> deltapz >> dp1x >> dp1y >> dp1z;
	      plane[ipl].z = z;
	      plane[ipl].nelem = nelem;
	      plane[ipl].cellwidth = cellwidth;
	      plane[ipl].spacing = spacing;
	      plane[ipl].xoffset = xoffset;
	      plane[ipl].scalex = scalex;
	      plane[ipl].x0 = x0;
	      plane[ipl].x1 = x1;
	      plane[ipl].x1 = x2;
	      plane[ipl].costheta = costheta;
	      plane[ipl].scaley = scaley;
	      plane[ipl].y0 = y0;
	      plane[ipl].y1 = y1;
	      plane[ipl].y2 = y2;
	      plane[ipl].sintheta = sintheta;
	      plane[ipl].resolution = resolution;
	      plane[ipl].p1x_w1 = p1x;
	      plane[ipl].p1y_w1 = p1y;
	      plane[ipl].p1z_w1 = p1z;
	      plane[ipl].deltapx = deltapx;
	      plane[ipl].deltapy = deltapy;
	      plane[ipl].deltapz = deltapz;
	      plane[ipl].dp1x = dp1x;
	      plane[ipl].dp1y = dp1y;
	      plane[ipl].dp1z = dp1z;
	      if(ipl>nChamberPlanes+nHodoPlanes){
		for(int k = 0; k<9; k++){
			iss >> deltaW_;
			plane[ipl].deltaW_[k] = deltaW_;
		}
	      }else{
		iss >> deltaW_;
		plane[ipl].deltaW_[0] = deltaW_;
	      }
	      plane[ipl].slope_max = costheta*TX_MAX+sintheta*TY_MAX;
	      plane[ipl].inter_max = costheta*X0_MAX+sintheta*Y0_MAX;
	      if(ipl%2==0 && ipl>1){
		double dslope = (plane[ipl].resolution+plane[ipl-1].resolution)/(plane[ipl].z-plane[ipl-1].z);
		double dinter = dslope*plane[ipl].z;
		plane[ipl].slope_max+= dslope;
		plane[ipl].inter_max+= dinter;
		plane[ipl-1].slope_max+= dslope;
		plane[ipl-1].inter_max+= dinter;
	      }
	      ipl++;
	}
	
	for(int i = 0; i<5; i++){
		int u_idx = i*6+5;
		if(i==0)u_idx = i*6+1;
		int x_idx = i*6+3;
		for(int j = 0; j<6; j++){
			int idx = i*6+j+1;
			plane[idx].z_mean = j%2==0 ? 0.5*(plane[idx].z+plane[idx+1].z):0.5*(plane[idx].z+plane[idx-1].z);
			
			plane[idx].v_win_fac1 = plane[idx].spacing*2*plane[u_idx].costheta;
			plane[idx].v_win_fac2 = plane[u_idx].costheta*TX_MAX;
			plane[idx].v_win_fac3 = plane[u_idx].sintheta*TY_MAX;
		}
		
		for(int j = 0; j<6; j++){
			int idx = i*6+j+1;
			plane[idx].u_win = fabs(0.5*plane[u_idx].scaley*plane[u_idx].sintheta) + TX_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].costheta) + TY_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].sintheta) + 2.*plane[u_idx].spacing + u_factor[i];
		}
		cout << u_idx << " " << plane[u_idx].u_win << " = " << fabs(0.5*plane[u_idx].scaley*plane[u_idx].sintheta) << " + " << TX_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].costheta) << " + " << TY_MAX*fabs((plane[u_idx].z_mean - plane[x_idx].z_mean)*plane[u_idx].sintheta) << " + " << 2.*plane[u_idx].spacing + u_factor[i] << endl;
		cout << " u costheta " << plane[u_idx].costheta << " u sintheta " << plane[u_idx].sintheta << " x_span " << plane[u_idx].scaley << " spacing " << plane[u_idx].spacing << " z plane_u " << plane[u_idx].z_mean << " z plane_x " << plane[x_idx].z_mean << endl;  
	}
	cout << "Geometry file read out" << endl;
	
	double wire_position[55][400];//Let's keep this: simpler, more robust
	for(int i = 1; i <= nChamberPlanes; ++i){
		//cout << plane[i].nelem << endl;
      		for(int j = 1; j <= plane[i].nelem; ++j){
          		double pos = (j - (plane[i].nelem+1.)/2.)*plane[i].spacing + plane[i].xoffset + plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].deltaW_[0];
			wire_position[i][j] = pos;
		}
	}
	for(int i = nChamberPlanes+1; i<=nChamberPlanes+nHodoPlanes; ++i){
		//cout << plane[i].nelem << endl;
	      	for(int j = 1; j <= plane[i].nelem; ++j){
          		double pos = plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].xoffset + (j - (plane[i].nelem+1)/2.)*plane[i].spacing + plane[i].deltaW_[0];
			wire_position[i][j] = pos;
		}
	}
	for(int i = nChamberPlanes+nHodoPlanes+1; i<=nChamberPlanes+nHodoPlanes+nPropPlanes; ++i){
		//cout << plane[i].nelem << endl;
	      	for(int j = 1; j <= plane[i].nelem; ++j){
          		int moduleID = 8 - int((j - 1)/8);
			//cout << moduleID << endl;
             		double pos = plane[i].x0*plane[i].costheta + plane[i].y0*plane[i].sintheta + plane[i].xoffset + (j - (plane[i].nelem+1)/2.)*plane[i].spacing + plane[i].deltaW_[moduleID];
			wire_position[i][j] = pos;
		}
		
	}
		
	TFile* dataFile = new TFile(inputFile.Data(), "READ");
	TTree* dataTree = 0;// = (TTree *)dataFile->Get("save");
	SRawEvent* rawEvent = new SRawEvent();
	//SQEvent_v1* event = new SQEvent_v1();
	//SQHitVector_v1* hitvec = new SQHitVector_v1();
	
	Int_t     _run_id;
 	Int_t     _spill_id;
	Int_t     _event_id;
   	UShort_t  _trigger;
	Int_t     _qie_presums[4];
	Int_t     _qie_turn_id = 0;
	Int_t     _qie_rf_id = 0;
	Int_t     _qie_rf_inte[33];

#ifdef E1039
	std::vector<SQHit*> hit_vec;
#endif
	
	if(e906data){
		dataTree = (TTree *)dataFile->Get("save");
		dataTree->SetBranchAddress("rawEvent", &rawEvent);
	}else{
#ifdef E1039
		//default option: e1039
		dataTree = (TTree *)dataFile->Get("T");
		dataTree->SetMakeClass(1); //this is necessary to get the tree to read the branch correctly
		dataTree->SetBranchStatus("*", 0);// this speeds up the loop on events

		dataTree->SetBranchStatus("DST.SQEvent._run_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._spill_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._event_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._trigger", 1);
		//dataTree->SetBranchStatus("DST.SQEvent._qie_presums[4]", 1);
		dataTree->SetBranchStatus("DST.SQEvent._qie_turn_id", 1);
		dataTree->SetBranchStatus("DST.SQEvent._qie_rf_id", 1);
		//dataTree->SetBranchStatus("DST.SQEvent._qie_rf_inte[33]", 1);
		
		dataTree->SetBranchAddress("DST.SQEvent._run_id", &_run_id);
		dataTree->SetBranchAddress("DST.SQEvent._spill_id", &_spill_id);
		dataTree->SetBranchAddress("DST.SQEvent._event_id", &_event_id);
		dataTree->SetBranchAddress("DST.SQEvent._trigger", &_trigger);
		//dataTree->SetBranchAddress("DST.SQEvent._qie_presums[4]", &_qie_presums);
		dataTree->SetBranchAddress("DST.SQEvent._qie_turn_id", &_qie_turn_id);
		dataTree->SetBranchAddress("DST.SQEvent._qie_rf_id", &_qie_rf_id);
		//dataTree->SetBranchAddress("DST.SQEvent._qie_rf_inte[33]", &_qie_rf_inte[33]);
		
		dataTree->SetBranchStatus("DST.SQHitVector._vector", 1);
		dataTree->SetBranchAddress("DST.SQHitVector._vector", &hit_vec);
#endif
	}
	int nEvtMax = dataTree->GetEntries();
	if(nEvtMax>EstnEvtMax)nEvtMax=EstnEvtMax;
	
	static gEvent host_gEvent[EstnEvtMax];
	
	cout << "unfolding " << nEvtMax <<" events" << endl;
	// loop on event: get RawEvent information and load it into gEvent
	for(int i = 0; i < nEvtMax; ++i) {
		if(i%1000==0)cout << i << "/" << nEvtMax <<  endl;
		dataTree->GetEntry(i);
		
		//cout<<"Converting "<<i<<"/"<<nEvtMax<<endl;
		if(e906data){
			host_gEvent[i].RunID = rawEvent->fRunID;
			host_gEvent[i].EventID = rawEvent->fEventID;
			host_gEvent[i].SpillID = rawEvent->fSpillID;
			host_gEvent[i].TriggerBits = rawEvent->fTriggerBits;
			host_gEvent[i].TargetPos = rawEvent->fTargetPos;
			host_gEvent[i].TurnID = rawEvent->fTurnID;
			host_gEvent[i].RFID = rawEvent->fRFID;
			for(int j=0; j<33; j++) {
				host_gEvent[i].Intensity[j] = rawEvent->fIntensity[j];
			}
			host_gEvent[i].TriggerEmu = rawEvent->fTriggerEmu;
			for(int k=0; k<4; k++) {
				host_gEvent[i].NRoads[k] = rawEvent->fNRoads[k];
			}
			for(int l=0; l<(nChamberPlanes+nHodoPlanes+nPropPlanes+1); l++) {
				host_gEvent[i].NHits[l] = rawEvent->fNHits[l];
			}
			host_gEvent[i].nAH = rawEvent->fAllHits.size();
			host_gEvent[i].nTH = rawEvent->fTriggerHits.size();
			for(int m=0; m<rawEvent->fAllHits.size(); m++) {
				host_gEvent[i].AllHits[m].index=(rawEvent->fAllHits[m]).index;
				host_gEvent[i].AllHits[m].detectorID=(rawEvent->fAllHits[m]).detectorID;
				host_gEvent[i].AllHits[m].elementID=(rawEvent->fAllHits[m]).elementID;
				host_gEvent[i].AllHits[m].tdcTime=(rawEvent->fAllHits[m]).tdcTime;
				host_gEvent[i].AllHits[m].driftDistance=(rawEvent->fAllHits[m]).driftDistance;
				host_gEvent[i].AllHits[m].pos=wire_position[(rawEvent->fAllHits[m]).detectorID][(rawEvent->fAllHits[m]).elementID];
				host_gEvent[i].AllHits[m].flag=(rawEvent->fAllHits[m]).flag;
			}
			for(int n=0; n<rawEvent->fTriggerHits.size(); n++) {
				host_gEvent[i].TriggerHits[n].index=(rawEvent->fTriggerHits[n]).index;
				host_gEvent[i].TriggerHits[n].detectorID=(rawEvent->fTriggerHits[n]).detectorID;
				host_gEvent[i].TriggerHits[n].elementID=(rawEvent->fTriggerHits[n]).elementID;
				host_gEvent[i].TriggerHits[n].tdcTime=(rawEvent->fTriggerHits[n]).tdcTime;
				host_gEvent[i].TriggerHits[n].driftDistance=(rawEvent->fTriggerHits[n]).driftDistance;
				host_gEvent[i].TriggerHits[n].pos=wire_position[(rawEvent->fAllHits[n]).detectorID][(rawEvent->fAllHits[n]).elementID];
				host_gEvent[i].TriggerHits[n].flag=(rawEvent->fTriggerHits[n]).flag;
			}
			// printouts for test
			//if(10000<rawEvent->fEventID&&rawEvent->fEventID<10050){
			//	printf("%d:\n ", rawEvent->fEventID);
			//	for(int l = 1; l<=nChamberPlanes; l++){
			//		printf("%d ", rawEvent->fNHits[l]);
			//	}printf("; %d\n", rawEvent->fAllHits.size());
			//	for(int m = 0; m<=50; m++){
			//		printf("%d, %1.3f;", (rawEvent->fAllHits[m]).detectorID, (rawEvent->fAllHits[m]).pos);
			//	}printf("\n");
			//}
		}else{
#ifdef E1039
			//Default option: e1039
			//if(_event_id<20)cout << " evt: " << _event_id << " nhits = " << hit_vec.size() << endl; 
			host_gEvent[i].RunID = _run_id;
			host_gEvent[i].SpillID = _spill_id;
			host_gEvent[i].EventID = _event_id;
			host_gEvent[i].TriggerBits = _trigger;
			//for(int k = 0; k<4; k++)host_gEvent[i].NRoads[k] = _qie_presums[k];
			host_gEvent[i].TurnID = _qie_turn_id;
			host_gEvent[i].RFID = _qie_rf_id;
			//for(int k = 0; k<33; k++)host_gEvent[i].Intensity[k] = _qie_rf_inte[k];
			
			for(int k = 0; k<nDetectors; k++)host_gEvent[i].NHits[k] = 0;//we will increment those in the vector hit loop
			int ntrighits = 0;
			host_gEvent[i].nAH = 0;//hit_vec.size();
			for(int m = 0; m<hit_vec.size(); m++){
				if(hit_vec[m]->get_detector_id()>54){
					//if(_event_id<20)cout << " dark photon plane hit! " << hit_vec[m]->get_detector_id() << endl;
					continue;
					//dark photon planes; I don't think we care about those for the purpose of online reconstruction... do we?
				}
				host_gEvent[i].nAH++;
				host_gEvent[i].NHits[hit_vec[m]->get_detector_id()]++;
				host_gEvent[i].AllHits[m].index=hit_vec[m]->get_hit_id();
				host_gEvent[i].AllHits[m].detectorID=hit_vec[m]->get_detector_id();
				host_gEvent[i].AllHits[m].elementID=hit_vec[m]->get_element_id();
				host_gEvent[i].AllHits[m].tdcTime=hit_vec[m]->get_tdc_time();
				host_gEvent[i].AllHits[m].driftDistance=fabs(hit_vec[m]->get_drift_distance());
				host_gEvent[i].AllHits[m].sign_mc=hit_vec[m]->get_drift_distance()/fabs(hit_vec[m]->get_drift_distance());
				host_gEvent[i].AllHits[m].pos=wire_position[hit_vec[m]->get_detector_id()][hit_vec[m]->get_element_id()];
				host_gEvent[i].AllHits[m].flag=(1<<hit_vec[m]->is_in_time());
				//if(host_gEvent[i].EventID<20)cout << " det " << host_gEvent[i].AllHits[m].detectorID << " elem " << host_gEvent[i].AllHits[m].elementID << " time " << host_gEvent[i].AllHits[m].tdcTime << " dd " << host_gEvent[i].AllHits[m].driftDistance << " pos " << host_gEvent[i].AllHits[m].pos << endl;
				if(hit_vec[m]->is_trigger_mask()){
					host_gEvent[i].TriggerHits[ntrighits].index=hit_vec[m]->get_hit_id();
					host_gEvent[i].TriggerHits[ntrighits].detectorID=hit_vec[m]->get_detector_id();
					host_gEvent[i].TriggerHits[ntrighits].elementID=hit_vec[m]->get_element_id();
					host_gEvent[i].TriggerHits[ntrighits].tdcTime=hit_vec[m]->get_tdc_time();
					host_gEvent[i].TriggerHits[ntrighits].driftDistance=fabs(hit_vec[m]->get_drift_distance());
					host_gEvent[i].TriggerHits[ntrighits].pos=wire_position[hit_vec[m]->get_detector_id()][hit_vec[m]->get_element_id()];
					host_gEvent[i].TriggerHits[ntrighits].flag=(1<<hit_vec[m]->is_in_time());
					ntrighits++;
				}
			}
			host_gEvent[i].nTH = ntrighits;
#endif
		}
	}
	cout << "loaded events" << endl;
	auto cp2 = std::chrono::system_clock::now();

	auto evt_prep = cp2-cp1;
	cout<<"Read/prepare events: "<<evt_prep.count()/1000000000.<<endl;

	// evaluate the total size of the gEvent array (and the SW array) for memory allocation 
	// (the memory cannot be dynamically allocated) 
	size_t NBytesAllEvent = EstnEvtMax * sizeof(gEvent);
	size_t NBytesAllOutputEvent = EstnEvtMax * sizeof(gOutputEvent);
	size_t NBytesAllPlanes =  nDetectors * sizeof(gPlane);
	size_t NBytesFitterTools = EstnEvtMax * sizeof(gStraightFitArrays);
	size_t NBytesStraightTrackBuilders = EstnEvtMax * sizeof(gStraightTrackBuilder);
	size_t NBytesFullTrackBuilders = EstnEvtMax * sizeof(gFullTrackBuilder);
	size_t NBytesKalmanFilterTools = EstnEvtMax * sizeof(gStraightFitArrays);
	

	cout << "Total size allocated on GPUs " << NBytesAllEvent+NBytesAllOutputEvent+NBytesAllPlanes+NBytesFitterTools << endl;
	cout << " input events: " << NBytesAllEvent << "; output events: " << NBytesAllOutputEvent << "; straight track builder tools: " << NBytesStraightTrackBuilders
	     << "; fitter tools: " << NBytesFitterTools << "; planes info: " << NBytesAllPlanes << endl;  
	
	gEvent *host_output_eR = (gEvent*)malloc(NBytesAllEvent);
	gOutputEvent *host_output_TKL = (gOutputEvent*)malloc(NBytesAllOutputEvent);
	
	// declaring gEvent objects for the device (GPU) to use.
	gEvent *device_gEvent;
	gOutputEvent *device_output_TKL;
	gPlane *device_gPlane;
	gStraightFitArrays *device_gFitArrays;
	gStraightTrackBuilder *device_gStraightTrackBuilder;
	gFullTrackBuilder *device_gFullTrackBuilder;
	gKalmanFitArrays *device_gKalmanFitArrays;

	//printDeviceStatus();

	// copy of data from host to device: evaluate operation time 
	// Allocating memory for GPU (pointer to allocated device ); check for errors in the process; stops the program if issues encountered
	gpuErrchk( hipMalloc((void**)&device_gEvent, NBytesAllEvent));
	gpuErrchk( hipMalloc((void**)&device_output_TKL, NBytesAllOutputEvent));
	//allocating the memory for the planes
	gpuErrchk( hipMalloc((void**)&device_gPlane, NBytesAllPlanes));
	gpuErrchk( hipMalloc((void**)&device_gFitArrays, NBytesFitterTools));
	gpuErrchk( hipMalloc((void**)&device_gStraightTrackBuilder, NBytesStraightTrackBuilders));

	std::size_t free_bytes;
	std::size_t total_bytes;

	CUDA_CHECK_STATUS(hipMemGetInfo(&free_bytes, &total_bytes));
    	cout << free_bytes << " / " << total_bytes << endl;
	
	// hipMemcpy(dst, src, count, kind): copies data between host and device:
	// dst: destination memory address; src: source memory address; count: size in bytes; kind: type of transfer
	gpuErrchk( hipMemcpy(device_gPlane, plane, NBytesAllPlanes, hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpy(device_gEvent, host_gEvent, NBytesAllEvent, hipMemcpyHostToDevice));
	auto cp3 = std::chrono::system_clock::now();

	auto cp_to_gpu = cp3-cp2;
	cout<<"Copy to GPU: "<<cp_to_gpu.count()/1000000000.<<endl;
		
	// now data is transfered in the device: kernel function for event reconstruction called;
	// note that the function call is made requesting a number of blocks and a number of threads per block
	// in practice we have as many threads total as number of events; 
	gkernel_eR<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent);
	
	// check status of device and synchronize;
	size_t nEvents = EstnEvtMax;
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	auto cp4 = std::chrono::system_clock::now();
	auto gpu_er = cp4-cp3;
	cout<<"GPU: event reducing: "<<gpu_er.count()/1000000000.<<endl;
	
	gKernel_XZ_YZ_tracking<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent, device_output_TKL, device_gStraightTrackBuilder, device_gFitArrays, device_gPlane);
	//gKernel_XZ_YZ_tracking_new<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent, device_output_TKL, device_gStraightTrackBuilder, device_gFitArrays, device_gPlane);
	
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	auto cp5 = std::chrono::system_clock::now();
	auto gpu_st = cp5-cp4;
	cout<<"GPU: straight tracking: "<<gpu_st.count()/1000000000.<<endl;

	//release here the memory for straight track builders and straight track fitters	
	hipFree(device_gStraightTrackBuilder);
	
	gpuErrchk( hipMalloc((void**)&device_gFullTrackBuilder, NBytesFullTrackBuilders));
	
	gKernel_GlobalTrack_building<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_gEvent, device_output_TKL, device_gFullTrackBuilder, device_gFitArrays, device_gPlane, 1);

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	gpuErrchk( hipMalloc((void**)&device_gKalmanFitArrays, NBytesKalmanFilterTools));
	
	auto cp6 = std::chrono::system_clock::now();
	auto gpu_gt = cp6-cp5;
	cout<<"GPU: global tracking: "<<gpu_gt.count()/1000000000.<<endl;

	//gKernel_GlobalTrack_KalmanFitting<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(device_output_TKL, device_gKalmanFitArrays, device_gPlane);

	//gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( hipDeviceSynchronize() );
	

	auto cp7 = std::chrono::system_clock::now();
	auto gpu_kf = cp7-cp6;
	cout<<"GPU: kalman filtering: "<<gpu_gt.count()/1000000000.<<endl;

	// data transfer from device to host
	gpuErrchk( hipMemcpy(host_output_eR, device_gEvent, NBytesAllEvent, hipMemcpyDeviceToHost));
	hipFree(device_gEvent);
	
	gpuErrchk( hipMemcpy(host_output_TKL, device_output_TKL, NBytesAllOutputEvent, hipMemcpyDeviceToHost));
	hipFree(device_output_TKL);

	auto cp8 = std::chrono::system_clock::now();
	auto cp_to_cpu = cp8-cp7;
	cout<<"Copy back to CPU: "<<cp_to_cpu.count()/1000000000.<<endl;

	ofstream out("OutputFile.txt");
	//Write in a file, 
	long tklctr = 0;
	long nEvtsTotal = 0;
	long nEvtsPass = 0;
	for(int n = 0; n<nEvtMax; n++){
		if(host_output_eR[n].nAH==0)continue;
		nEvtsTotal++;
		if(host_output_eR[n].HasTooManyHits)continue;
		nEvtsPass++;
		out<<n<<" "<< host_output_eR[n].nAH <<" "<< host_output_TKL[n].nTracklets<<endl;
		tklctr+= host_output_TKL[n].nTracklets;
		for(int k = 1; k<=nDetectors; k++ ){
			out << host_output_eR[n].NHits[k] << " ";
		}out<<endl;
		
		for(int k = 0; k<host_output_eR[n].nAH; k++ ){
			out << host_output_eR[n].AllHits[k].detectorID << " " << host_output_eR[n].AllHits[k].elementID << " " << host_output_eR[n].AllHits[k].driftDistance*host_output_eR[n].AllHits[k].sign_mc << endl;
		}
		
		for(int k = 0; k<host_output_TKL[n].nTracklets; k++ ){
			if(isnan(host_output_TKL[n].AllTracklets[k].x0))host_output_TKL[n].AllTracklets[k].x0 = -1000;
			if(isnan(host_output_TKL[n].AllTracklets[k].y0))host_output_TKL[n].AllTracklets[k].y0 = -1000;
			if(isnan(host_output_TKL[n].AllTracklets[k].invP))host_output_TKL[n].AllTracklets[k].invP = -1.0;
			if(isnan(host_output_TKL[n].AllTracklets[k].tx))host_output_TKL[n].AllTracklets[k].tx = -1.0;
			if(isnan(host_output_TKL[n].AllTracklets[k].ty))host_output_TKL[n].AllTracklets[k].ty = -1.0;
			out << host_output_TKL[n].AllTracklets[k].stationID << " " << host_output_TKL[n].AllTracklets[k].x0 << " " << host_output_TKL[n].AllTracklets[k].y0 << " " << host_output_TKL[n].AllTracklets[k].tx << " " << host_output_TKL[n].AllTracklets[k].ty << " " << host_output_TKL[n].AllTracklets[k].invP << " " << host_output_TKL[n].AllTracklets[k].nXHits+host_output_TKL[n].AllTracklets[k].nUHits+host_output_TKL[n].AllTracklets[k].nVHits << endl;
			//if(n<100 && host_output_TKL[n].nTracklets>1)cout << n << " " << host_output_TKL[n].AllTracklets[k].stationID << " " << host_output_TKL[n].AllTracklets[k].nXHits<< " " <<host_output_TKL[n].AllTracklets[k].nUHits<< " " <<host_output_TKL[n].AllTracklets[k].nVHits << endl;
			for(int l = 0; l<host_output_TKL[n].AllTracklets[k].nXHits+host_output_TKL[n].AllTracklets[k].nUHits+host_output_TKL[n].AllTracklets[k].nVHits; l++){
				out << host_output_TKL[n].AllTracklets[k].hits[l].detectorID << " " << host_output_TKL[n].AllTracklets[k].hits[l].elementID << " " << host_output_TKL[n].AllTracklets[k].hits[l].driftDistance*host_output_TKL[n].AllTracklets[k].hitsign[l] << " " << host_output_TKL[n].AllTracklets[k].hits[l].pos << endl;
			}
		}
		
	}

	cout << tklctr << " total tracks reconstructed" << endl;
	cout << nEvtsPass << " evts with low enough number of hits on " << nEvtsTotal << " events total." << endl; 
	//auto end_kernel = std::chrono::system_clock::now();


	delete rawEvent;

	//TFile* outFile = new TFile(outputFile.Data(), "RECREATE");
	//ORoutput_tree* output = new ORoutput_tree();
	//for(int i = 0; i < nEvtMax; ++i) {
	//	output->Clear();
	//	for(int k = 1; k<=nDetectors; k++ )output->fNhitsReduced[k] = host_output_eR[i].NHits[k];
	//	//output->Write();
	//}
	//output->Write();

	auto cp9 = std::chrono::system_clock::now();
	auto write_output = cp9-cp8;
	cout<<"Write Output: "<<write_output.count()/1000000000.<<endl;

	// printing the time required for all operations
	auto end = std::chrono::system_clock::now();
	auto overall = end - start;
	cout<<"Total time: "<<overall.count()/1000000000.<<endl;
		
	return 0;
}
